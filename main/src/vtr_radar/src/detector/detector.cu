#include "hip/hip_runtime.h"
#include "vtr_radar/detector/detector.cuh"
#include "vtr_radar/data_types/point.hpp"
// #include <opencv2/imgcodecs.hpp>
// #include <opencv2/highgui.hpp>
namespace vtr {
namespace radar {
  __global__ void computeMeans_kernel(float* raw_scan, int rows, int cols, int min_col, int max_col, double* means) {
    int row_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row_index >= rows)
      return;
    means[row_index] = 0;
    for (int j = min_col; j < max_col; ++j) {
      means[row_index] = means[row_index] + raw_scan[row_index * cols + j];
    }
    means[row_index] = means[row_index] / (max_col - min_col);
    return;
  }
  __global__ void modifiedCACFAR_kernel(float* raw_scan,
                                     char* th_matrix,
                                     double* means,
                                     int min_col,
                                     int max_col,
                                     int rows,
                                     int cols,
                                     int w2,
                                     int guard,
                                     double threshold,
                                     double threshold2,
                                     double threshold3) {
    int row_index = blockIdx.x * blockDim.x + threadIdx.x;
    int col_index = blockIdx.y * blockDim.y + threadIdx.y;
    if (row_index >= rows || col_index < min_col || col_index >= max_col)
      return;

    double left = 0;
    double right = 0;
    for (int k = -w2 - guard; k < -guard; ++k) left = left + raw_scan[row_index * cols + col_index + k];
    for (int k = guard + 1; k <= w2 + guard; ++k) right = right + raw_scan[row_index * cols + col_index + k];
    // (statistic) estimate of clutter power
    // const double stat = (left + right) / (2 * w2);
    const double stat = fmax(left, right) / w2;  // GO-CFAR
    const float thres = threshold * stat + threshold2 * means[row_index] + threshold3;
    if (raw_scan[row_index * cols + col_index] > thres) 
      th_matrix[row_index * cols + col_index] = 255;
    return;
  }

  void cudaModifiedCACFAR(CudaMem& gpu_mem,
                            double minr,
                            double maxr,
                            int w2,
                            int guard,
                            double range_offset,
                            double th,
                            double th2,
                            double th3,
                            const cv::Mat &raw_scan, const float &res, 
                            const std::vector<int64_t> &azimuth_times,
                            const std::vector<double> &azimuth_angles,
                            pcl::PointCloud<PointT> &pointcloud) {

    pointcloud.clear();
    int rows = raw_scan.rows;
    const int cols = raw_scan.cols;
    
    int mincol = minr / res + w2 + guard + 1;
    if (mincol > cols || mincol < 0) mincol = 0;
    
    int maxcol = maxr / res - w2 - guard;
    if (maxcol > cols || maxcol < 0) maxcol = cols;
    

    dim3 dim_block(16, 16);
    dim3 dim_grid;
    dim_grid.x = (rows + dim_block.x - 1) / dim_block.x;
    dim_grid.y = (cols + dim_block.y - 1) / dim_block.y;

    const unsigned int block_size = 256;
    const unsigned int num_blocks = (rows + block_size - 1) / block_size;

    computeMeans_kernel<<<num_blocks, block_size>>>(gpu_mem.raw_scan_device, rows, cols, mincol, maxcol, gpu_mem.means_device);
    hipDeviceSynchronize();
    // call the kernel

    modifiedCACFAR_kernel<<<dim_grid, dim_block>>>(gpu_mem.raw_scan_device, 
                          gpu_mem.th_matrix_device, gpu_mem.means_device,
                          mincol, maxcol, rows, cols, w2, guard, th, th2, th3);
    hipDeviceSynchronize();
    gpu_mem.fromGpu();
    // std::cerr << rows << "   " << cols << std::endl;
    // cv::imshow("Display window", gpu_mem.th_mat);
    // int k = cv::waitKey();
    // #pragma omp declare reduction(merge_points : std::vector<Point3D> : omp_out.insert( \
    //       omp_out.end(), omp_in.begin(), omp_in.end())) initializer(omp_priv = decltype(omp_orig)(omp_orig.size()))
    // #pragma omp parallel for num_threads(num_threads_) reduction(merge_points : raw_points)
    for (int row = 0; row < rows; ++row) {
      float peak_points = 0;
      int num_peak_points = 0;
      const double azimuth = azimuth_angles[row];
      const double time = azimuth_times[row];
      pcl::PointCloud<PointT> polar_time;

      for (int col = mincol; col < maxcol; ++col) {
        if (gpu_mem.th_mat.at<uchar>(row, col) == 255) {
          peak_points += col;
          num_peak_points += 1;
        } else if (num_peak_points > 0) {
          PointT p;
          p.rho = res * peak_points / num_peak_points + range_offset;
          p.phi = azimuth;
          p.theta = 0;
          p.timestamp = time;
          polar_time.push_back(p);
          peak_points = 0;
          num_peak_points = 0;
        }
      }
      pointcloud.insert(pointcloud.end(), polar_time.begin(), polar_time.end());
    }
    return;
  }


template 
void cudaModifiedCACFAR<PointWithInfo>(CudaMem& gpu_mem,
                            double minr,
                            double maxr,
                            int w2,
                            int guard,
                            double range_offset,
                            double th,
                            double th2,
                            double th3,
                            const cv::Mat &raw_scan, const float &res, 
                            const std::vector<int64_t> &azimuth_times,
                            const std::vector<double> &azimuth_angles,
                            pcl::PointCloud<PointWithInfo> &pointcloud);
}  // namespace radar
}  // namespace vtr