#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision$
// $Date$
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 
#include <cudpp_globals.h>
#include "cudpp_radixsort.h"
#include "cta/scan_cta.cu"
#include <cudpp.h>
#include <stdio.h>

#include <cudpp_util.h>
#include <math.h>
#include "sharedmem.h"


#ifdef __DEVICE_EMULATION__
#define __EMUSYNC __syncthreads()
#else
#define __EMUSYNC
#endif

/**
 * @file
 * sort_cta.cu
 * 
 * @brief CUDPP CTA-level sort routines
 */

/** \addtogroup cudpp_cta 
* @{
*/

/** @name Radix Sort Functions
* @{
*/


typedef unsigned int uint;

/**
 * @brief Flips bits of single-precision floating-point number (parameterized by doFlip)
 * 
 *  flip a float for sorting
 *  finds SIGN of fp number.
 *  if it's 1 (negative float), it flips all bits
 *  if it's 0 (positive float), it flips the sign only
 * @param[in] f floating-point input (passed as unsigned int)
 * @see floatUnflip
**/

template <bool doFlip>
__device__ uint floatFlip(uint f)
{
    if (doFlip)
    {
        uint mask = -int(f >> 31) | 0x80000000;
        return f ^ mask;
    }
    else
        return f;
}

/**
 * @brief Reverses bit-flip of single-precision floating-point number (parameterized by doFlip)
 * 
 * flip a float back (invert FloatFlip)
 *  signed was flipped from above, so:
 *  if sign is 1 (negative), it flips the sign bit back
 *  if sign is 0 (positive), it flips all bits back
 * @param[in] f floating-point input (passed as unsigned int)
 * @see floatFlip
**/
template <bool doFlip>
__device__ uint floatUnflip(uint f)
{
    if (doFlip)
    {
        uint mask = ((f >> 31) - 1) | 0x80000000;
        return f ^ mask;
    }
    else
        return f;
}

/**
 * @brief Scans one warp quickly, optimized for 32-element warps, using shared memory
 * 
 * Scans each warp in parallel ("warp-scan"), one element per thread.
 * uses 2 numElements of shared memory per thread (64 numElements per warp)
 * 
 * @param[in] val Elements per thread to scan
 * @param[in,out] sData
**/
template<class T, int maxlevel>
__device__ T scanwarp(T val, volatile T* sData)
{
    // The following is the same as 2 * WARP_SIZE * warpId + threadInWarp = 
    // 64*(threadIdx.x >> 5) + (threadIdx.x & (WARP_SIZE - 1))
    int idx = 2 * threadIdx.x - (threadIdx.x & (WARP_SIZE - 1));
    sData[idx] = 0;
    idx += WARP_SIZE;
    T t = sData[idx] = val;          __EMUSYNC;

#ifdef __DEVICE_EMULATION__             
        t = sData[idx -  1]; __EMUSYNC; 
        sData[idx] += t;       __EMUSYNC;
        t = sData[idx -  2];   __EMUSYNC; 
        sData[idx] += t;       __EMUSYNC;
        t = sData[idx -  4];   __EMUSYNC; 
        sData[idx] += t;       __EMUSYNC;
        t = sData[idx -  8];   __EMUSYNC; 
        sData[idx] += t;       __EMUSYNC;
        t = sData[idx - 16];   __EMUSYNC; 
        sData[idx] += t;       __EMUSYNC;
#else
        if (0 <= maxlevel) { sData[idx] = t = t + sData[idx - 1]; } __EMUSYNC;
        if (1 <= maxlevel) { sData[idx] = t = t + sData[idx - 2]; } __EMUSYNC;
        if (2 <= maxlevel) { sData[idx] = t = t + sData[idx - 4]; } __EMUSYNC;
        if (3 <= maxlevel) { sData[idx] = t = t + sData[idx - 8]; } __EMUSYNC;
        if (4 <= maxlevel) { sData[idx] = t = t + sData[idx -16]; } __EMUSYNC;
#endif          
        return sData[idx] - val;  // convert inclusive -> exclusive
}

/**
 * @brief Scans 4*CTA_SIZE unsigned ints in a block
 *
 * scan4 scans 4*CTA_SIZE numElements in a block (4 per
 * thread), using a warp-scan algorithm
 * 
 * @param[in] idata 4-vector of integers to scan
**/
__device__ uint4 scan4(uint4 idata)
{    
    extern  __shared__  uint ptr[];
    
    uint idx = threadIdx.x;

    uint4 val4 = idata;
    uint sum[3];
    sum[0] = val4.x;
    sum[1] = val4.y + sum[0];
    sum[2] = val4.z + sum[1];
    
    uint val = val4.w + sum[2];
    
    val = scanwarp<uint, 4>(val, ptr);
    __syncthreads();

    if ((idx & (WARP_SIZE - 1)) == WARP_SIZE - 1)
    {
        ptr[idx >> 5] = val + val4.w + sum[2];
    }
    __syncthreads();

#ifndef __DEVICE_EMULATION__
    if (idx < WARP_SIZE)
#endif
    {
        ptr[idx] = scanwarp<uint, 2>(ptr[idx], ptr);
    }
    __syncthreads();

    val += ptr[idx >> 5];

    val4.x = val;
    val4.y = val + sum[0];
    val4.z = val + sum[1];
    val4.w = val + sum[2];      
        
    return val4;
}

/**
 * @brief Computes output position for each thread given predicate; trues come first then falses
 * 
 * Rank is the core of the radix sort loop.  Given a predicate, it
 * computes the output position for each thread in an ordering where all
 * True threads come first, followed by all False threads. 
 * This version handles 4 predicates per thread; hence, "rank4".
 *
 * @param[in] preds true/false values for each of the 4 elements in this thread
 *
 * @todo is the description of "preds" correct?
**/
template <int ctasize>
__device__ uint4 rank4(uint4 preds)
{
    uint4 address = scan4(preds);  

    __shared__ uint numtrue;
    if (threadIdx.x == ctasize-1)
    {
        numtrue = address.w + preds.w;
    }
    __syncthreads();

    uint4 rank;
    uint idx = threadIdx.x << 2;
    rank.x = (preds.x) ? address.x : numtrue + idx   - address.x;
    rank.y = (preds.y) ? address.y : numtrue + idx + 1 - address.y;
    rank.z = (preds.z) ? address.z : numtrue + idx + 2 - address.z;
    rank.w = (preds.w) ? address.w : numtrue + idx + 3 - address.w;     
                
    return rank;
}

/**
 * @brief Sorts one block
 *
 * Uses rank to sort one bit at a time: Sorts a block according
 * to bits startbit -> nbits + startbit
 * @param[in,out] key
 * @param[in,out] value
**/
template<uint nbits, uint startbit>
__device__ void radixSortBlock(uint4 &key, uint4 &value)
{
    extern __shared__ uint sMem1[];
    for(uint shift = startbit; shift < (startbit + nbits); ++shift)
    {        
        uint4 lsb;
        lsb.x = !((key.x >> shift) & 0x1);
        lsb.y = !((key.y >> shift) & 0x1);
        lsb.z = !((key.z >> shift) & 0x1);
        lsb.w = !((key.w >> shift) & 0x1); 

        uint4 r = rank4<256>(lsb);

#if 1
        // This arithmetic strides the ranks across 4 SORT_CTA_SIZE regions
        sMem1[(r.x & 3) * SORT_CTA_SIZE + (r.x >> 2)] = key.x;
        sMem1[(r.y & 3) * SORT_CTA_SIZE + (r.y >> 2)] = key.y;
        sMem1[(r.z & 3) * SORT_CTA_SIZE + (r.z >> 2)] = key.z;
        sMem1[(r.w & 3) * SORT_CTA_SIZE + (r.w >> 2)] = key.w; 
        __syncthreads();

        // The above allows us to read without 4-way bank conflicts:
        key.x = sMem1[threadIdx.x];
        key.y = sMem1[threadIdx.x +     SORT_CTA_SIZE];
        key.z = sMem1[threadIdx.x + 2 * SORT_CTA_SIZE];
        key.w = sMem1[threadIdx.x + 3 * SORT_CTA_SIZE];

        __syncthreads();

        sMem1[(r.x & 3) * SORT_CTA_SIZE + (r.x >> 2)] = value.x;
        sMem1[(r.y & 3) * SORT_CTA_SIZE + (r.y >> 2)] = value.y;
        sMem1[(r.z & 3) * SORT_CTA_SIZE + (r.z >> 2)] = value.z;
        sMem1[(r.w & 3) * SORT_CTA_SIZE + (r.w >> 2)] = value.w;
        __syncthreads();

        value.x = sMem1[threadIdx.x];
        value.y = sMem1[threadIdx.x +     SORT_CTA_SIZE];
        value.z = sMem1[threadIdx.x + 2 * SORT_CTA_SIZE];
        value.w = sMem1[threadIdx.x + 3 * SORT_CTA_SIZE];
#else
        sMem1[r.x] = key.x;
        sMem1[r.y] = key.y;
        sMem1[r.z] = key.z;
        sMem1[r.w] = key.w;
        __syncthreads();

        // This access has 4-way bank conflicts
        key = sMem[threadIdx.x];

        __syncthreads();

        sMem1[r.x] = value.x;
        sMem1[r.y] = value.y;
        sMem1[r.z] = value.z;
        sMem1[r.w] = value.w;
        __syncthreads();

        value = sMem[threadIdx.x];
#endif

        __syncthreads();
    }
}

/**
 * @brief Sorts one block. Key-only version.
 *
 * Uses rank to sort one bit at a time: Sorts a block according
 * to bits startbit -> nbits + startbit
 * @param[in,out] key
**/

template<uint nbits, uint startbit>
__device__ void radixSortBlockKeysOnly(uint4 &key)
{
    extern __shared__ uint sMem1[];
    for(uint shift = startbit; shift < (startbit + nbits); ++shift)
    {                   
        uint4 lsb;
        lsb.x = !((key.x >> shift) & 0x1);
        lsb.y = !((key.y >> shift) & 0x1);
        lsb.z = !((key.z >> shift) & 0x1);
        lsb.w = !((key.w >> shift) & 0x1);

        uint4 r = rank4<256>(lsb);

#if 1
        // This arithmetic strides the ranks across 4 CTA_SIZE regions
        sMem1[(r.x & 3) * SORT_CTA_SIZE + (r.x >> 2)] = key.x;
        sMem1[(r.y & 3) * SORT_CTA_SIZE + (r.y >> 2)] = key.y;
        sMem1[(r.z & 3) * SORT_CTA_SIZE + (r.z >> 2)] = key.z;
        sMem1[(r.w & 3) * SORT_CTA_SIZE + (r.w >> 2)] = key.w;
        __syncthreads();

        // The above allows us to read without 4-way bank conflicts:
        key.x = sMem1[threadIdx.x];
        key.y = sMem1[threadIdx.x +     SORT_CTA_SIZE];
        key.z = sMem1[threadIdx.x + 2 * SORT_CTA_SIZE];
        key.w = sMem1[threadIdx.x + 3 * SORT_CTA_SIZE];
#else
        sMem1[r.x] = key.x;
        sMem1[r.y] = key.y;
        sMem1[r.z] = key.z;
        sMem1[r.w] = key.w;
        __syncthreads();

        // This access has 4-way bank conflicts
        key = sMem[threadIdx.x];
#endif

        __syncthreads();
    }
}

/** @} */ // end radix sort functions
/** @} */ // end cudpp_cta
