#include "hip/hip_runtime.h"
// ***************************************************************
//  cuDPP -- CUDA Data Parallel Primitives library
//  -------------------------------------------------------------
//  $Revision: 3505 $
//  $Date: 2007-07-06 09:26:06 -0700 (Fri, 06 Jul 2007) $
//  -------------------------------------------------------------
// This source code is distributed under the terms of license.txt in
// the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
* @file
* segmented_scan_app.cu
*
* @brief CUDPP application-level scan routines
*/

/** \defgroup cudpp_app CUDPP Application-Level API
* The CUDPP Application-Level API contains functions
* that run on the host CPU and invoke GPU routines in 
* the CUDPP \link cudpp_kernel Kernel-Level API\endlink. 
* Application-Level API functions are used by
* CUDPP \link publicInterface Public Interface\endlink
* functions to implement CUDPP's core functionality.
* @{
*/

/** @name Segmented Scan Functions
* @{
*/

#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_plan.h"
#include "kernel/segmented_scan_kernel.cu"
#include "kernel/vector_kernel.cu"


#include <cutil.h>
#include <cstdlib>
#include <cstdio>
#include <assert.h>

/** @brief Perform recursive scan on arbitrary size arrays
*
* This is the CPU-side workhorse function of the segmented scan
* engine. This function invokes the CUDA kernels which perform the
* segmented scan on individual blocks.
*
* Scans of large arrays must be split (possibly recursively) into a
* hierarchy of block scans, where each block is scanned by a single
* CUDA thread block. At each recursive level of the
* segmentedScanArrayRecursive first invokes a kernel to scan all blocks of
* that level, and if the level has more than one block, it calls
* itself recursively. On returning from each recursive level, the
* total sum of each block from the level below is added to all
* elements of the first segment of the corresponding block in this
* level.
*
* Template parameter T is the data type of the input data.
* Template parameter op is the binary operator of the segmented scan.
* Template parameter isBackward specifies whether the direction is backward
* (not implemented). It is forward if it is false.
* Template parameter isExclusive specifies whether the segmented scan
* is exclusive (true) or inclusive (false).
*
* @param[out] d_out The output array for the segmented scan results
* @param[in] d_idata The input array to be scanned
* @param[in] d_iflags The input flags vector which specifies the
* segments. The first element of a segment is marked by a 1 in the
* corresponding position in d_iflags vector. All other elements of
* d_iflags is 0.
* @param[out] d_blockSums Array of arrays of per-block sums (one
* array per recursive level, allocated
* by allocScanStorage())
* @param[out] d_blockFlags Array of arrays of per-block OR-reductions
* of flags (one array per recursive level, allocated by
* allocScanStorage())
* @param[out] d_blockIndices Array of arrays of per-block
* min-reductions of indices (one array per recursive level, allocated
* by allocSegmentedScanStorage()). An index for a particular position \c i in
* a block is calculated as - if \c d_iflags[i] is set then it is the
* 1-based index of that position (i.e if \c d_iflags[10] is set then
* index is \c 11) otherwise the index is \c INT_MAX (the identity
* element of a min operator)
* @param[in] numElements The number of elements in the array to scan
* @param[in] level The current recursive level of the scan
*/
template <class T, CUDPPOperator op, bool isBackward, bool isExclusive, bool doShiftFlagsLeft>
void segmentedScanArrayRecursive(T                  *d_out, 
                                 const T            *d_idata, 
                                 const unsigned int *d_iflags,
                                 T                  **d_blockSums,
                                 unsigned int       **d_blockFlags,
                                 unsigned int       **d_blockIndices,
                                 int                numElements,
                                 int                level)
{
    unsigned int numBlocks = 
        max(1, (int)ceil((double)numElements / 
        ((double)SEGSCAN_ELTS_PER_THREAD * CTA_SIZE)));

    // This is the number of elements per block that the 
    // CTA level API is aware of
    unsigned int numEltsPerBlock = CTA_SIZE * 2;

    // Space to store flags - we need two sets. One gets modified and the
    // other doesn't
    unsigned int flagSpace = numEltsPerBlock * sizeof(unsigned int);

    // Space to store indices
    unsigned int idxSpace = numEltsPerBlock * sizeof(unsigned int);

    // Total shared memory space
    unsigned int sharedMemSize = 
        sizeof(T) * (numEltsPerBlock) + idxSpace + flagSpace;

    // setup execution parameters
    dim3  grid(max(1, numBlocks), 1, 1);
    dim3  threads(CTA_SIZE, 1, 1);

    // make sure there are no CUDA errors before we start
    CUT_CHECK_ERROR("segmentedScanArrayRecursive before kernels");

    bool fullBlock = (numElements == 
        (numBlocks * SEGSCAN_ELTS_PER_THREAD * CTA_SIZE));    

    bool sm12OrBetterHw;
    hipDeviceProp_t deviceProp;
    int dev;
    CUDA_SAFE_CALL(hipGetDevice(&dev));
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev));
    if (deviceProp.minor >= 2)
        sm12OrBetterHw = true;
    else
        sm12OrBetterHw = false;

    unsigned int traitsCode = 0;
    if (numBlocks > 1)  traitsCode |= 1;
    if (fullBlock)      traitsCode |= 2;
    if (sm12OrBetterHw) traitsCode |= 4;

    switch(traitsCode)
    {
    case 0: // single block, single row, non-full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, false, false,
                       false> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements, 0, 0, 0);
        break;
    case 1: // multi block, single row, non-full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, false, true,
                       false> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements,
            d_blockSums[level], d_blockFlags[level],
            d_blockIndices[level]);
        break;
    case 2: // single block, single row, full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, true, false,
                       false> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements, 0, 0, 0);
        break;
    case 3: // multi block, single row, full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, true, true,
                       false> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements,
            d_blockSums[level], d_blockFlags[level],
            d_blockIndices[level]);
        break;
    case 4: // single block, single row, non-full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, false, false,
                       true> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements, 0, 0, 0);
        break;
    case 5: // multi block, single row, non-full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, false, true,
                       true> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements,
            d_blockSums[level], d_blockFlags[level],
            d_blockIndices[level]);
        break;
    case 6: // single block, single row, full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, true, false,
                       true> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements, 0, 0, 0);
        break;
    case 7: // multi block, single row, full last block
        segmentedScan4<T, SegmentedScanTraits<T, op, isBackward, isExclusive, doShiftFlagsLeft, true, true,
                       true> >
            <<< grid, threads, sharedMemSize >>>
            (d_out, d_idata, d_iflags, numElements,
            d_blockSums[level], d_blockFlags[level],
            d_blockIndices[level]);
        break;
    }

    CUT_CHECK_ERROR("segmentedScanArrayRecursive after block level scans");

    if (numBlocks > 1)
    {
        // After scanning all the sub-blocks, we are mostly done. But
        // now we need to take all of the last values of the
        // sub-blocks and segment scan those. This will give us a new value
        // that must be sdded to the first segment of each block to get 
        // the final results.
        segmentedScanArrayRecursive<T, op, isBackward, false, false>
            ((T*)d_blockSums[level], (const T*)d_blockSums[level], 
            d_blockFlags[level], (T **)d_blockSums,
            d_blockFlags, d_blockIndices,
            numBlocks, level + 1);

        if (isBackward)
        {
            if (fullBlock)
                vectorSegmentedAddUniformToRight4<T, op, true><<<grid, threads>>>
                (d_out, d_blockSums[level], d_blockIndices[level], 
                numElements, 0, 0);
            else
                vectorSegmentedAddUniformToRight4<T, op, false><<<grid, threads>>>
                (d_out, d_blockSums[level], d_blockIndices[level], 
                numElements, 0, 0);
        }
        else
        {
            if (fullBlock)
                vectorSegmentedAddUniform4<T, op, true><<<grid, threads>>>
                (d_out, d_blockSums[level], d_blockIndices[level], 
                numElements, 0, 0);
            else
                vectorSegmentedAddUniform4<T, op, false><<<grid, threads>>>
                (d_out, d_blockSums[level], d_blockIndices[level], 
                numElements, 0, 0);
        }

        CUT_CHECK_ERROR("vectorSegmentedAddUniform4");
    }
}

#ifdef __cplusplus
extern "C" 
{
#endif

    // file scope
    /** @brief Allocate intermediate block sums, block flags and block
    *        indices arrays in a CUDPPSegmentedScanPlan class.
    *
    * Segmented scans of large arrays must be split (possibly
    * recursively) into a hierarchy of block segmented scans, where each
    * block is scanned by a single CUDA thread block. At each recursive
    * level of the scan, we need an array in which to store the total
    * sums of all blocks in that level. Also at this level we have two
    * more arrays - one which contains the OR-reductions of flags of all
    * blocks at that level and the second which contains the
    * min-reductions of indices of all blocks at that levels This
    * function computes the amount of storage needed and allocates it.
    *
    * @param[in] plan Pointer to CUDPPSegmentedScanPlan object containing segmented scan
    * options and number of elements, which is used to compute storage
    * requirements.
    */
    void allocSegmentedScanStorage(CUDPPSegmentedScanPlan *plan)
    {
        plan->m_numEltsAllocated = plan->m_numElements;

        size_t numElts = plan->m_numElements;

        size_t level = 0;

        do
        {       
            size_t numBlocks = 
                max(1, (unsigned int)ceil
                ((double)numElts / 
                ((double)SEGSCAN_ELTS_PER_THREAD * CTA_SIZE)));
            if (numBlocks > 1)
            {
                level++;
            }
            numElts = numBlocks;
        } while (numElts > 1);

        size_t elementSize = 0;

        switch(plan->m_config.datatype)
        {
        case CUDPP_INT:
            plan->m_blockSums = (void**) malloc(level * sizeof(int*));
            elementSize = sizeof(int);
            break;
        case CUDPP_UINT:
            plan->m_blockSums = (void**) malloc(level * sizeof(unsigned int*));
            elementSize = sizeof(unsigned int);
            break;
        case CUDPP_FLOAT:
            plan->m_blockSums = (void**) malloc(level * sizeof(float*));
            elementSize = sizeof(float);
            break;
        default:
            break;
        }

        plan->m_blockFlags = 
            (unsigned int**) malloc(level * sizeof(unsigned int*));
        plan->m_blockIndices = 
            (unsigned int**) malloc(level * sizeof(unsigned int*));

        plan->m_numLevelsAllocated = level;
        numElts = plan->m_numElements;

        level = 0;

        do
        {       
            size_t numBlocks = 
                max(1, 
                (unsigned int)ceil((double)numElts / 
                ((double)SEGSCAN_ELTS_PER_THREAD * CTA_SIZE)));
            if (numBlocks > 1) 
            {
                CUDA_SAFE_CALL(hipMalloc((void**) &(plan->m_blockSums[level]),
                    numBlocks * elementSize));
                CUDA_SAFE_CALL(hipMalloc((void**) &(plan->m_blockFlags[level]),
                    numBlocks * sizeof(unsigned int)));
                CUDA_SAFE_CALL(hipMalloc((void**) &(plan->m_blockIndices[level]),  
                    numBlocks * sizeof(unsigned int)));
                level++;
            }
            numElts = numBlocks;
        } while (numElts > 1);

        CUT_CHECK_ERROR("allocSegmentedScanStorage");
    }

    /** @brief Deallocate intermediate block sums, block flags and block
    *        indices arrays in a CUDPPSegmentedScanPlan class.
    *
    * These arrays must have been allocated by allocSegmentedScanStorage(),
    * which is called by the constructor of CUDPPSegmentedScanPlan.
    *
    * @param[in] plan CUDPPSegmentedScanPlan class initialized by its constructor.
    */
    void freeSegmentedScanStorage(CUDPPSegmentedScanPlan *plan)
    {
        for (unsigned int i = 0; i < plan->m_numLevelsAllocated; i++)
        {
            hipFree(plan->m_blockSums[i]);
            hipFree(plan->m_blockFlags[i]);
            hipFree(plan->m_blockIndices[i]);
        }

        CUT_CHECK_ERROR("freeSegmentedScanStorage");

        free((void**)plan->m_blockSums);
        free((void**)plan->m_blockFlags);
        free((void**)plan->m_blockIndices);

        plan->m_blockSums = 0;
        plan->m_blockFlags = 0;
        plan->m_blockIndices = 0;
        plan->m_numEltsAllocated = 0;
        plan->m_numLevelsAllocated = 0;
    }

    /** @brief Dispatch function to perform a scan (prefix sum) on an
    * array with the specified configuration.
    *
    * This is the dispatch routine which calls segmentedScanArrayRecursive() with 
    * appropriate template parameters and arguments to achieve the scan as 
    * specified in \a plan. 
    * 
    * @param[in]  numElements The number of elements to scan
    * @param[in]  plan        Segmented Scan configuration (plan), initialized 
    *                         by CUDPPSegmentedScanPlan constructor
    * @param[in]  d_idata     The input array
    * @param[in]  d_iflags    The input flags array

    * @param[out] d_out    The output array of segmented scan results
    */
    void cudppSegmentedScanDispatch (void                         *d_out, 
        const void                   *d_idata,
        const unsigned int           *d_iflags,
        int                          numElements,
        const CUDPPSegmentedScanPlan *plan
        )
    {    
        if (CUDPP_OPTION_EXCLUSIVE & plan->m_config.options)
        {
            if (CUDPP_OPTION_BACKWARD & plan->m_config.options)
            {
                switch(plan->m_config.datatype)
                {
                case CUDPP_INT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<int, CUDPP_MAX, true, true, true>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<int, CUDPP_ADD, true, true, true>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<int, CUDPP_MULTIPLY, true, true, true>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<int, CUDPP_MIN, true, true, true>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                case CUDPP_UINT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MAX, true, true, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_ADD, true, true, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MULTIPLY, true, true, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MIN, true, true, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                        (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                        plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                case CUDPP_FLOAT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<float, CUDPP_MAX, true, true, true>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<float, CUDPP_ADD, true, true, true>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<float, CUDPP_MULTIPLY, true, true, true>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<float, CUDPP_MIN, true, true, true>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                default:
                    break;
                }
            }
            else
            {
                switch(plan->m_config.datatype)
                {
                case CUDPP_INT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<int, CUDPP_MAX, false, true, false>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<int, CUDPP_ADD, false, true, false>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<int, CUDPP_MULTIPLY, false, true, false>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<int, CUDPP_MIN, false, true, false>
                            ((int *)d_out, (int *)d_idata, d_iflags, 
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                case CUDPP_UINT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MAX, false, true, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_ADD, false, true, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MULTIPLY, false, true, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MIN, false, true, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags, 
                        (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                        plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                case CUDPP_FLOAT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<float, CUDPP_MAX, false, true, false>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<float, CUDPP_ADD, false, true, false>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<float, CUDPP_MULTIPLY, false, true, false>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<float, CUDPP_MIN, false, true, false>
                            ((float *)d_out, (float *)d_idata, d_iflags, 
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                default:
                    break;
                }
            }
        }
        else
        {
            if (CUDPP_OPTION_BACKWARD & plan->m_config.options)
            {
                switch(plan->m_config.datatype)
                {
                case CUDPP_INT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<int, CUDPP_MAX, true, false, true>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<int, CUDPP_ADD, true, false, true>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<int, CUDPP_MULTIPLY, true, false, true>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<int, CUDPP_MIN, true, false, true>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                break;
                case CUDPP_UINT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MAX, true, false, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_ADD, true, false, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MULTIPLY, true, false, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MIN, true, false, true>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                case CUDPP_FLOAT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<float, CUDPP_MAX, true, false, true>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<float, CUDPP_ADD, true, false, true>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<float, CUDPP_MULTIPLY, true, false, true>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<float, CUDPP_MIN, true, false, true>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                default:
                    break;
                }
            }
            else
            {
                switch(plan->m_config.datatype)
                {
                case CUDPP_INT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<int, CUDPP_MAX, false, false, false>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<int, CUDPP_ADD, false, false, false>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<int, CUDPP_MULTIPLY, false, false, false>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<int, CUDPP_MIN, false, false, false>
                            ((int *)d_out, (int *)d_idata, d_iflags,
                            (int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                break;
                case CUDPP_UINT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MAX, false, false, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_ADD, false, false, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MULTIPLY, false, false, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<unsigned int, CUDPP_MIN, false, false, false>
                            ((unsigned int *)d_out, (unsigned int *)d_idata, d_iflags,
                            (unsigned int **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                case CUDPP_FLOAT:
                    switch(plan->m_config.op)
                    {
                    case CUDPP_MAX:
                        segmentedScanArrayRecursive<float, CUDPP_MAX, false, false, false>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_ADD:
                        segmentedScanArrayRecursive<float, CUDPP_ADD, false, false, false>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MULTIPLY:
                        segmentedScanArrayRecursive<float, CUDPP_MULTIPLY, false, false, false>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    case CUDPP_MIN:
                        segmentedScanArrayRecursive<float, CUDPP_MIN, false, false, false>
                            ((float *)d_out, (float *)d_idata, d_iflags,
                            (float **)plan->m_blockSums, plan->m_blockFlags,
                            plan->m_blockIndices, numElements, 0);
                        break;
                    default:
                        break;
                    }
                    break;
                default:
                    break;
                }
            }
        }
    }

#ifdef __cplusplus
}
#endif

/** @} */ // end segmented scan functions
/** @} */ // end cudpp_app
