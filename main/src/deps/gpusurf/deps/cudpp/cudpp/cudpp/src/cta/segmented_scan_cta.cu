#include "hip/hip_runtime.h"
// ***************************************************************
//  cuDPP -- CUDA Data Parallel Primitives library
//  -------------------------------------------------------------
//  $Revision: 3512 $
//  $Date: 2007-07-06 15:39:28 -0700 (Fri, 06 Jul 2007) $
//  -------------------------------------------------------------
// This source code is distributed under the terms of license.txt in
// the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
* @file
* segmented_scan_cta.cu
*
* @brief CUDPP CTA-level scan routines
*/

/** \addtogroup cudpp_cta 
* @{
*/

/** @name Segmented scan Functions
* @{
*/

#include "cta/scan_cta.cu"
#include <cudpp_globals.h>
#include <math.h>
#include <cstdio>

/**
  * @brief Template class containing compile-time parameters to the segmented scan functions
  *
  * SegmentedScanTraits is passed as a template parameter to all segmented scan functions.  By 
  * using these compile-time functions we can enable generic code while 
  * maintaining the highest performance.  This is crucial for the performance 
  * of low-level workhorse algorithms like segmented scan.
  *
  * @param T The datatype of the segmented scan
  * @param oper The ::CUDPPOperator to use for the segmented scan (add, max, etc.)
  * @param unroll True if scan inner loops should be unrolled
  * @param sums True if each block should write it's sum to the d_blockSums array (false for single-block scans)
  * @param backward True if this is a backward scan, False if this is a forward scan
  * @param fullBlock True if all blocks in this scan are full (CTA_SIZE * SCAN_ELEMENTS_PER_THREAD elements)
  * @param exclusivity True for exclusive scans, false for inclusive scans
*/
template <class T, CUDPPOperator oper, bool backward, bool exclusivity, 
          bool doShiftFlags, bool fullBlock, bool sums, bool sm12OrBetter>
class SegmentedScanTraits
{
public:
    //! Returns true if this is a backward scan
    static __device__ bool isBackward()   { return backward;   }
    //! Returns true if this is an exclusive scan
    static __device__ bool isExclusive()  { return exclusivity; }
    //! Returns true if this scan needs to shift flags to the left. This is only needed for the first level scan
    //! in a multi-block scan
    static __device__ bool shiftFlags() { return doShiftFlags; }
    //! Returns true if this is a full scan -- all blocks process CTA_SIZE * SCAN_ELEMENTS_PER_THREAD elements
    static __device__ bool isFullBlock() { return fullBlock;        }
    //! Returns true if this scan writes the sum of each block to the d_blockSums array (multi-block scans)
    static __device__ bool writeSums() { return sums; }
    //! Returns true if we are sm12 or better hardware
    static __device__ bool isSM12OrBetter() { return sm12OrBetter; }


    //! The operator function used for segmented scan
    static __device__ T op(const T a, const T b)
    {
        return Operator<T, oper>::op(a, b);
    }  

    //! The identity value used by segmented scan
    static __device__ T identity() { return Operator<T, oper>::identity(); }
};

/**
* @brief Handles loading input s_data from global memory to shared memory 
* (vec4 version)
*
* Load a chunk of 8*blockDim.x elements from global memory into a 
* shared memory array.  Each thread loads two T4 elements (where
* T4 is, e.g. int4 or float4), computes the segmented scan of those two vec4s in 
* thread local arrays (in registers), and writes the two total sums of the
* vec4s into shared memory, where they will be cooperatively scanned with 
* the other partial sums by all threads in the CTA.
*
* @param[out] s_odata The output (shared) memory array
* @param[out] threadScan0 Intermediate per-thread partial sums array 1
* @param[out] threadScan1 Intermediate per-thread partial sums array 2
* @param[out] threadFlag Intermediate array which holds 8 flags as follows
* Temporary register threadFlag0[4] - the flags for the first 4 elements read
* Temporary register threadFlag1[4] - the flags for the second 4 elements read
* Temporary register threadScanFlag0[4] - the inclusive OR-scan for the flags in threadFlag0[4]
* Temporary register threadScanFlag1[4] - the inclusive OR-scan for the flags in threadFlag1[4]
* We storing the 16 flags 32 bits of threadFlag
* Bits 0...3 contains threadFlag0[0]...threadFlag0[3]
* Bits 4...7 contains threadFlag1[0]...threadFlag1[3]
* Bits 8...11 contains threadScanFlag0[0]...threadScanFlag0[3]
* Bits 11...15 contains threadScanFlag1[0]...threadScanFlag1[3]
* @param[out] s_oflags Output (shared) memory array of segment head flags
* @param[out] s_oindices Output (shared) memory array of indices. If a flag for a position (1-based)
*                        is set then index for that position is the position, 0 otherwise.
* @param[in] d_idata The input (device) memory array
* @param[in] d_iflags The input (device) memory array of segment head flags
* @param[in] numElements The number of elements in the array being scanned
* @param[in] iDataOffset the offset of the input array in global memory for this 
* thread block
* @param[out] ai The shared memory address for the thread's first element 
* (returned for reuse)
* @param[out] bi The shared memory address for the thread's second element 
* (returned for reuse)
* @param[out] aiDev The device memory address for this thread's first element 
* (returned for reuse)
* @param[out] biDev The device memory address for this thread's second element 
* (returned for reuse)
*/
template <class T, typename traits>
inline __device__ 
void 
loadForSegmentedScanSharedChunkFromMem4(
                                        T *s_odata,
                                        T threadScan0[4],
                                        T threadScan1[4],
                                        unsigned int& threadFlag,
                                        unsigned int* s_oflags,
                                        unsigned int* s_oindices,
                                        const T *d_idata,
                                        const unsigned int *d_iflags,
                                        int numElements,
                                        int iDataOffset,
                                        int& ai, 
                                        int& bi, 
                                        int& aiDev, 
                                        int& biDev
                                        )
{
    int thid = threadIdx.x;

    aiDev = iDataOffset + threadIdx.x;
    biDev = aiDev + blockDim.x;

    ai = thid;
    bi = thid + blockDim.x;

// #ifndef __DEVICE_EMULATION__
    bool isLastBlock = (blockIdx.x == (gridDim.x-1));
// #endif

    // convert to 4-vector
    typename typeToVector<T,4>::Result* iData = (typename typeToVector<T,4>::Result*)d_idata;
    typename typeToVector<unsigned int,4>::Result* iFlags = (typename typeToVector<unsigned,4>::Result*)d_iflags;
    typename typeToVector<T,4>::Result tempData;

    uint4 tempFlag;

    unsigned int gIndex = (aiDev) * 4;

#ifdef __DEVICE_EMULATION__
    if (traits::shiftFlags())
    {
        if (traits::isFullBlock() || (gIndex+4) < numElements)
        {   
            tempFlag.x = d_iflags[gIndex+1];
            tempFlag.y = d_iflags[gIndex+2];
            tempFlag.z = d_iflags[gIndex+3];
            tempFlag.w = d_iflags[gIndex+4];
        }    
        else
        {
            tempFlag.x = ((gIndex+1) < numElements) ? d_iflags[gIndex+1] : 0;
            tempFlag.y = ((gIndex+2) < numElements) ? d_iflags[gIndex+2] : 0;
            tempFlag.z = ((gIndex+3) < numElements) ? d_iflags[gIndex+3] : 0;
            tempFlag.w = 0;
        }   
    }
    else
    {
        if (traits::isFullBlock() || (gIndex+3) < numElements)
        {
            tempFlag = iFlags[aiDev];
        }
        else
        {
            tempFlag.x = (gIndex < numElements) ? d_iflags[gIndex] : 0;
            tempFlag.y = ((gIndex+1) < numElements) ? d_iflags[gIndex+1] : 0;
            tempFlag.z = ((gIndex+2) < numElements) ? d_iflags[gIndex+2] : 0;
            tempFlag.w = 0;
        }
    }
#else

    if (traits::shiftFlags() && traits::isSM12OrBetter())
    {
        if (traits::isFullBlock() || (gIndex+4) <  numElements)
        {
            tempFlag.x = d_iflags[gIndex+1];
            tempFlag.y = d_iflags[gIndex+2];
            tempFlag.z = d_iflags[gIndex+3];
            tempFlag.w = d_iflags[gIndex+4];
        }
        else
        {
            tempFlag.x = ((gIndex+1) < numElements) ? d_iflags[gIndex+1] : 0;
            tempFlag.y = ((gIndex+2) < numElements) ? d_iflags[gIndex+2] : 0;
            tempFlag.z = ((gIndex+3) < numElements) ? d_iflags[gIndex+3] : 0;
            tempFlag.w = 0;
        }   
    }
    else
    {
        tempFlag = iFlags[aiDev];

        if (isLastBlock && !traits::isFullBlock())
        {
            if (gIndex >= numElements) tempFlag.x = 0;
            if ((gIndex+1) >= numElements) tempFlag.y = 0;
            if ((gIndex+2) >= numElements) tempFlag.z = 0;
            if ((gIndex+3) >= numElements) tempFlag.w = 0;
        }
    }

    // Pad values beyond numElements with identity elements 

    if (traits::shiftFlags() && !traits::isSM12OrBetter())
    {
        if (ai == 0)
        {
            unsigned int t = (iDataOffset + blockDim.x)*(4);
            if (isLastBlock)
                s_oflags[blockDim.x-1] = (t < numElements) ? d_iflags[t] : 0;
            else
                s_oflags[blockDim.x-1] = d_iflags[t];
        }
        else
        {
            s_oflags[ai-1] = tempFlag.x;
        }
        
        // Inside an if but the if should be evaluated at compile time
        __syncthreads();

        tempFlag.x = tempFlag.y;
        tempFlag.y = tempFlag.z;
        tempFlag.z = tempFlag.w;
        tempFlag.w = s_oflags[ai];

        // Do I need a __syncthreads here - I don't think so
    }
#endif

    // Store the first 4 flags in threadFlag[0]...threadFlag[3]
    threadFlag = 0;
    threadFlag |= tempFlag.x;
    threadFlag |= (tempFlag.y << 1);
    threadFlag |= (tempFlag.z << 2);
    threadFlag |= (tempFlag.w << 3);
#ifdef __DEVICE_EMULATION__
    if (traits::isFullBlock() || (gIndex+3) < numElements)
    {
        tempData = iData[aiDev];
    }
    else
    {
        tempData.x = (gIndex < numElements) ? d_idata[gIndex] : traits::identity();
        tempData.y = ((gIndex+1) < numElements) ? d_idata[gIndex+1] : traits::identity();
        tempData.z = ((gIndex+2) < numElements) ? d_idata[gIndex+2] : traits::identity();
        tempData.w = traits::identity();        
    }
#else
    // Read 4 data
    // Pad values beyond numElements with identity elements
    tempData = iData[aiDev];
    if (isLastBlock && !traits::isFullBlock())
    {
        if (gIndex     >= numElements) tempData.x = traits::identity();
        if ((gIndex+1) >= numElements) tempData.y = traits::identity();
        if ((gIndex+2) >= numElements) tempData.z = traits::identity();
        if ((gIndex+3) >= numElements) tempData.w = traits::identity();
    }
#endif

    // Computed inclusive segmented scan and store result in
    // threadScan0
    if (traits::isBackward())
    {
        threadScan0[3] = tempData.w;
        threadScan0[2] = 
            traits::op(tempData.z, tempFlag.z ? traits::identity() : threadScan0[3]);
        threadScan0[1] = 
            traits::op(tempData.y, tempFlag.y ? traits::identity() : threadScan0[2]);
        threadScan0[0] = s_odata[ai] = 
            traits::op(tempData.x, tempFlag.x ? traits::identity() : threadScan0[1]);
    }
    else
    {
        threadScan0[0] = tempData.x;
        threadScan0[1] = 
            traits::op(tempData.y, tempFlag.y ? traits::identity() : threadScan0[0]);
        threadScan0[2] = 
            traits::op(tempData.z, tempFlag.z ? traits::identity() : threadScan0[1]);
        threadScan0[3] = s_odata[ai] =
            traits::op(tempData.w, tempFlag.w ? traits::identity() : threadScan0[2]);
    }

    unsigned int indexVec[4];

    if (traits::isBackward())
    {
        // Compute 4 indices. The logic is if a flag in this position
        // is 1 then the index is set to the 1-based position (i.e if
        // gIndex is 10 then index is set to 11). If flag is 0 then 
        // index is set to the identity element for max which is 
        // 0
        indexVec[0] = (gIndex + 1 + 0) * tempFlag.x;
        indexVec[1] = (gIndex + 1 + 1) * tempFlag.y;
        indexVec[2] = (gIndex + 1 + 2) * tempFlag.z;
        indexVec[3] = (gIndex + 1 + 3) * tempFlag.w;
    }
    else
    {
        // Compute 4 indices. The logic is if a flag in this position
        // is 1 then the index is set to the 1-based position (i.e if
        // gIndex is 10 then index is set to 11). If flag is 0 then 
        // index is set to the identity element for min which is 
        // UINT_MAX
        indexVec[0] = 
            (gIndex + 1 + 0) * tempFlag.x + (1 - tempFlag.x) * UINT_MAX;
        indexVec[1] = 
            (gIndex + 1 + 1) * tempFlag.y + (1 - tempFlag.y) * UINT_MAX;
        indexVec[2] = 
            (gIndex + 1 + 2) * tempFlag.z + (1 - tempFlag.z) * UINT_MAX;
        indexVec[3] = 
            (gIndex + 1 + 3) * tempFlag.w + (1 - tempFlag.w) * UINT_MAX;
    }

    unsigned int m_index;

    if (traits::isBackward())
    {
        // Compute maximum of 4 indices
        m_index = 
            max(max(max(indexVec[0], indexVec[1]), indexVec[2]), indexVec[3]);
    }
    else
    {
        // Compute minimum of 4 indices
        m_index = 
            min(min(min(indexVec[0], indexVec[1]), indexVec[2]), indexVec[3]);
    }

    // Store the minimum/maximum index in shared memory
    s_oindices[ai] = m_index;

    // Store inclusive OR-scan of 4 flags read in threadFlag[8]...threadFlag[11]
    if (traits::isBackward())
    {
        threadFlag |= 
            ((tempFlag.w | tempFlag.z | tempFlag.y | tempFlag.x ) << 8);
        threadFlag |= ((tempFlag.w | tempFlag.z | tempFlag.y) << 9);
        threadFlag |= ((tempFlag.w | tempFlag.z) << 10);
        threadFlag |= (tempFlag.w << 11);
    }
    else
    {
        threadFlag |= (tempFlag.x << 8);
        threadFlag |= ((tempFlag.x | tempFlag.y) << 9);
        threadFlag |= ((tempFlag.x | tempFlag.y | tempFlag.z) << 10);
        threadFlag |= 
            ((tempFlag.x | tempFlag.y | tempFlag.z | tempFlag.w) << 11);
    }

    // Store the OR-reduce of 4 flags in shared memory
    if (traits::isBackward())
        s_oflags[ai] = ((threadFlag >> 8) & 1);
    else
        s_oflags[ai] = ((threadFlag >> 11) & 1);

    gIndex = biDev * 4;

#ifdef __DEVICE_EMULATION__ 
    if (traits::shiftFlags())
    {
        if (traits::isFullBlock() || (gIndex+4) < numElements)
        {   
            tempFlag.x = d_iflags[gIndex+1];
            tempFlag.y = d_iflags[gIndex+2];
            tempFlag.z = d_iflags[gIndex+3];
            if (isLastBlock && (bi==((blockDim.x<<1)-1)))
                tempFlag.w = 0;
            else
                tempFlag.w = d_iflags[gIndex+4];
        }    
        else
        {
            tempFlag.x = ((gIndex+1) < numElements) ? d_iflags[gIndex+1] : 0;
            tempFlag.y = ((gIndex+2) < numElements) ? d_iflags[gIndex+2] : 0;
            tempFlag.z = ((gIndex+3) < numElements) ? d_iflags[gIndex+3] : 0;
            tempFlag.w = 0;
        } 
    }
    else
    { 
        if (traits::isFullBlock() || (gIndex+3) < numElements)
        {
            tempFlag = iFlags[biDev];
        }
        else
        {
            tempFlag.x = (gIndex < numElements) ? d_iflags[gIndex] : 0;
            tempFlag.y = ((gIndex+1) < numElements) ? d_iflags[gIndex+1] : 0;
            tempFlag.z = ((gIndex+2) < numElements) ? d_iflags[gIndex+2] : 0;
            tempFlag.w = 0;
        }
    }
#else
    // Read 4 flags
    // Pad values beyond numElements with identity elements
    if (traits::shiftFlags() && traits::isSM12OrBetter())
    {
        if (traits::isFullBlock() || (gIndex+4) < numElements)
        {   
            tempFlag.x = d_iflags[gIndex+1];
            tempFlag.y = d_iflags[gIndex+2];
            tempFlag.z = d_iflags[gIndex+3];
            if (isLastBlock && (bi==((blockDim.x<<1)-1)))
                tempFlag.w = 0;
            else
                tempFlag.w = d_iflags[gIndex+4];
        }
        else
        {
            tempFlag.x = ((gIndex+1) < numElements) ? d_iflags[gIndex+1] : 0;
            tempFlag.y = ((gIndex+2) < numElements) ? d_iflags[gIndex+2] : 0;
            tempFlag.z = ((gIndex+3) < numElements) ? d_iflags[gIndex+3] : 0;
            tempFlag.w = 0;
        }   
    }
    else
    {
        tempFlag = iFlags[biDev];

        if (isLastBlock && !traits::isFullBlock())
        {
            if (gIndex >= numElements) tempFlag.x = 0;
            if ((gIndex+1) >= numElements) tempFlag.y = 0;
            if ((gIndex+2) >= numElements) tempFlag.z = 0;
            if ((gIndex+3) >= numElements) tempFlag.w = 0;
        }
    } 

    if (traits::shiftFlags() && !traits::isSM12OrBetter())
    {
        if (bi == blockDim.x)
        {
            if (isLastBlock)
                s_oflags[(blockDim.x<<1)-1] = 0;
            else
                s_oflags[(blockDim.x<<1)-1] =
                    d_iflags[(iDataOffset + (blockDim.x<<1))*4];
        }
        else
        {
            s_oflags[bi-1] = tempFlag.x;
        }
        
        // Inside an if but the if should be evaluated at compile time
        __syncthreads();

        tempFlag.x = tempFlag.y;
        tempFlag.y = tempFlag.z;
        tempFlag.z = tempFlag.w;
        tempFlag.w = s_oflags[bi];

        // Do I need a __syncthreads here - I don't think so
    }
#endif

    // Store the first 4 flags in threadFlag[4]...threadFlag[7]
    threadFlag |= (tempFlag.x << 4);
    threadFlag |= (tempFlag.y << 5);
    threadFlag |= (tempFlag.z << 6);
    threadFlag |= (tempFlag.w << 7);

#ifdef __DEVICE_EMULATION__
    if (traits::isFullBlock() || (gIndex+3) < numElements)
    {
        tempData = iData[biDev];
    }
    else
    {
        tempData.x = (gIndex < numElements) ? d_idata[gIndex] : traits::identity();
        tempData.y = ((gIndex+1) < numElements) ? d_idata[gIndex+1] : traits::identity();
        tempData.z = ((gIndex+2) < numElements) ? d_idata[gIndex+2] : traits::identity();
        tempData.w = traits::identity();
    }
#else
    // Read 4 data
    // Pad values beyond numElements with identity elements
    tempData = iData[biDev];

    // Pad values beyond numElements with identity elements 
    if (isLastBlock && !traits::isFullBlock())
    {
        if (gIndex     >= numElements) tempData.x = traits::identity();
        if ((gIndex+1) >= numElements) tempData.y = traits::identity();
        if ((gIndex+2) >= numElements) tempData.z = traits::identity();
        if ((gIndex+3) >= numElements) tempData.w = traits::identity();
    }
#endif

    // Computed inclusive segmented scan and store result in
    // threadScan1
    if (traits::isBackward())
    {
        threadScan1[3] = tempData.w;
        threadScan1[2] = 
            traits::op(tempData.z, tempFlag.z ? traits::identity() : threadScan1[3]);
        threadScan1[1] = 
            traits::op(tempData.y, tempFlag.y ? traits::identity() : threadScan1[2]);
        threadScan1[0] = s_odata[bi] = 
            traits::op(tempData.x, tempFlag.x ? traits::identity() : threadScan1[1]);
    }
    else
    {
        threadScan1[0] = tempData.x;
        threadScan1[1] = 
            traits::op(tempData.y, tempFlag.y ? traits::identity() : threadScan1[0]);
        threadScan1[2] = 
            traits::op(tempData.z, tempFlag.z ? traits::identity() : threadScan1[1]);
        threadScan1[3] = s_odata[bi] = 
            traits::op(tempData.w, tempFlag.w ? traits::identity() : threadScan1[2]);
    }

    if (traits::isBackward())
    {
        // Compute 4 indices. Thelogic is if a flag in this position
        // is 1 then the index is set to the 1-based position (i.e if
        // gIndex is 10 then index is set to 11). If flag is 0 then 
        // index is set to the identity element for max which is 
        // 0
        indexVec[0] = 
            (gIndex + 1 + 0) * tempFlag.x;
        indexVec[1] = 
            (gIndex + 1 + 1) * tempFlag.y;
        indexVec[2] = 
            (gIndex + 1 + 2) * tempFlag.z;
        indexVec[3] = 
            (gIndex + 1 + 3) * tempFlag.w;
    }
    else
    {
        // Compute 4 indices. Thelogic is if a flag in this position
        // is 1 then the index is set to the 1-based position (i.e if
        // gIndex is 10 then index is set to 11). If flag is 0 then 
        // index is set to the identity element for min which is 
        // INT_MAX
        indexVec[0] = 
            (gIndex + 1 + 0) * tempFlag.x + (1 - tempFlag.x) * UINT_MAX;
        indexVec[1] = 
            (gIndex + 1 + 1) * tempFlag.y + (1 - tempFlag.y) * UINT_MAX;
        indexVec[2] = 
            (gIndex + 1 + 2) * tempFlag.z + (1 - tempFlag.z) * UINT_MAX;
        indexVec[3] = 
            (gIndex + 1 + 3) * tempFlag.w + (1 - tempFlag.w) * UINT_MAX;
    }

    if (traits::isBackward())
    {
        // Compute the minimum of 4 indices
        m_index = 
            max(max(max(indexVec[0], indexVec[1]), indexVec[2]), indexVec[3]);
    }
    else
    {
        // Compute the minimum of 4 indices
        m_index = 
            min(min(min(indexVec[0], indexVec[1]), indexVec[2]), indexVec[3]);
    }

    // Store the minimum index in shared memory
    s_oindices[bi] = m_index;

    // Store inclusive OR-scan of 4 flags read in threadFlag[12]...threadFlag[15]
    if (traits::isBackward())
    {
        threadFlag |= 
            ((tempFlag.w | tempFlag.z | tempFlag.y | tempFlag.x) << 12);
        threadFlag |= 
            ((tempFlag.w | tempFlag.z | tempFlag.y) << 13);
        threadFlag |= 
            ((tempFlag.w | tempFlag.z) << 14);
        threadFlag |= 
            (tempFlag.w << 15);
    }
    else
    {
        threadFlag |= (tempFlag.x << 12);
        threadFlag |= ((tempFlag.x | tempFlag.y) << 13);
        threadFlag |= ((tempFlag.x | tempFlag.y | tempFlag.z) << 14);
        threadFlag |= 
            ((tempFlag.x | tempFlag.y | tempFlag.z | tempFlag.w) << 15);
    }

    // Store the OR-reduce of 4 flags in shared memory
    if (traits::isBackward())
        s_oflags[bi] = ((threadFlag >> 12) & 1);
    else
        s_oflags[bi] = ((threadFlag >> 15) & 1);

    __syncthreads();
}



/**
* @brief Handles storing result s_data from shared memory to global memory 
* (vec4 version)
*
* Store a chunk of 8*blockDim.x elements from shared memory into a 
* device memory array.  Each thread stores reads two elements from shared
* memory, adds them while respecting segment bouldaries, to the intermediate
* sums computed in loadForSegmentedScanSharedChunkFromMem4(), and writes two T4 
* elements (where T4 is, e.g. int4 or float4) to global memory.
*
* @param[out] d_odata The output (device) memory array
* @param[out] threadScan0 Intermediate per-thread partial sums array 1
* (contents computed in loadForSegmentedScanSharedChunkFromMem4())
* @param[in] threadScan1 Intermediate per-thread partial sums array 2
* (contents computed in loadForSegmentedScanSharedChunkFromMem4())
* @param[in] threadFlag Various flags that loadForSegmentedScanSharedChunkFromMem4()
*            needs to pass
* @param[in] s_idata The input (shared) memory array
* @param[in] numElements The number of elements in the array being scanned
* @param[in] oDataOffset the offset of the output array in global memory 
* for this thread block
* @param[in] ai The shared memory address for the thread's first element 
* (computed in loadForSegmentedScanSharedChunkFromMem4())
* @param[in] bi The shared memory address for the thread's second element 
* (computed in loadForSegmentedScanSharedChunkFromMem4())
* @param[in] aiDev The device memory address for this thread's first element 
* (computed in loadForSegmentedScanSharedChunkFromMem4())
* @param[in] biDev The device memory address for this thread's second element 
* (computed in loadForSegmentedScanSharedChunkFromMem4())
*/
template <class T, class traits>
inline __device__ 
void storeForSegmentedScanSharedChunkToMem4(T *d_odata,
                                            T threadScan0[4],
                                            T threadScan1[4],
                                            unsigned int threadFlag,
                                            T *s_idata, 
                                            unsigned int numElements,
                                            int oDataOffset,
                                            int ai, 
                                            int bi, 
                                            int aiDev, 
                                            int biDev
                                            )
{
    bool isLastBlock = (blockIdx.x == (gridDim.x-1));

    // Convert to 4-vector
    typename typeToVector<T,4>::Result tempData;
    typename typeToVector<T,4>::Result* oData = (typename typeToVector<T,4>::Result*)d_odata;

    T temp;
    // To make it exclusive
    if (traits::isBackward())
    {
        temp = s_idata[ai+1];
    }
    else
    {
        if (ai == 0)
            temp = traits::identity(); 
        else
            temp = s_idata[ai-1]; 
    }

    // perform a 4-tuple wide segmented scan (either exclusive
    // or inclusive)
    if (traits::isExclusive())
    {
        if (traits::isBackward())
        {
            tempData.x = 
                traits::op(((threadFlag >> 8) & 1) ? traits::identity() : temp,
                       ((threadFlag >> 0) & 1) ? traits::identity() : threadScan0[1]);
            tempData.y = 
                traits::op(((threadFlag >> 9) & 1) ? traits::identity() : temp,
                       ((threadFlag >> 1) & 1) ? traits::identity() : threadScan0[2]); 
            tempData.z = 
                traits::op(((threadFlag >> 10) & 1) ? traits::identity() : temp,
                       ((threadFlag >> 2) & 1) ? traits::identity() : threadScan0[3]); 
            tempData.w = 
                ((threadFlag >> 11) & 1) ? traits::identity() : temp;
        }
        else
        {
            tempData.x = 
                ((threadFlag >> 8) & 1) ? traits::identity() : temp;
            tempData.y = 
                traits::op(((threadFlag >> 9) & 1) ? traits::identity() : temp,
                       ((threadFlag >> 1) & 1) ? traits::identity() : threadScan0[0]); 
            tempData.z = 
                traits::op(((threadFlag >> 10) & 1) ? traits::identity() : temp,
                       ((threadFlag >> 2) & 1) ? traits::identity() : threadScan0[1]); 
            tempData.w = 
                traits::op(((threadFlag >> 11) & 1) ? traits::identity() : temp,
                       ((threadFlag >> 3) & 1) ? traits::identity() : threadScan0[2]);
        }
    }
    else
    {
            tempData.x =
                traits::op(((threadFlag >> 8) & 1) ? traits::identity() : temp, 
                       threadScan0[0]);
            tempData.y = 
                traits::op(((threadFlag >> 9) & 1) ? traits::identity() : temp, 
                       threadScan0[1]);
            tempData.z =
                traits::op(((threadFlag >> 10) & 1) ? traits::identity() : temp, 
                       threadScan0[2]);
            tempData.w =
                traits::op(((threadFlag >> 11) & 1) ? traits::identity() : temp, 
                       threadScan0[3]);
    }

    // write results to global memory
    if (isLastBlock && !traits::isFullBlock())
    {            
        unsigned int i = aiDev * 4;
        if (i < numElements) {d_odata[i] = tempData.x;}
        if ((i+1) < numElements) {d_odata[i+1] = tempData.y;}
        if ((i+2) < numElements) {d_odata[i+2] = tempData.z;}
        if ((i+3) < numElements) {d_odata[i+3] = tempData.w;}
    }
    else
    {   
        oData[aiDev] = tempData; 
    }

    // To make it inclusive
    if (traits::isBackward())
    {
        if (bi == ((blockDim.x<<1)-1))
            temp = traits::identity(); 
        else
            temp = s_idata[bi+1]; 
    }
    else
    {
        temp = s_idata[bi-1]; 
    }

    // perform a 4-tuple wide segmented scan (either exclusive
    // or inclusive)
    if (traits::isExclusive())
    {
        if (traits::isBackward())
        {
            tempData.x = 
                traits::op(((threadFlag >> 12) & 1) ? traits::identity() : temp,
                       ((threadFlag >>  4) & 1) ? traits::identity() : threadScan1[1]);
            tempData.y = 
                traits::op(((threadFlag >> 13) & 1) ? traits::identity() : temp,
                       ((threadFlag >>  5) & 1) ? traits::identity() : threadScan1[2]);
            tempData.z = 
                traits::op(((threadFlag >> 14) & 1) ? traits::identity() : temp,
                       ((threadFlag >>  6) & 1) ? traits::identity() : threadScan1[3]);
            tempData.w = ((threadFlag >> 15) & 1) ? traits::identity() : temp;
        }
        else
        {
            tempData.x = 
                ((threadFlag >> 12) & 1) ? traits::identity() : temp;
            tempData.y = 
                traits::op(((threadFlag >> 13) & 1) ? traits::identity() : temp,
                       ((threadFlag >>  5) & 1) ? traits::identity() : threadScan1[0]);
            tempData.z = 
                traits::op(((threadFlag >> 14) & 1) ? traits::identity() : temp,
                       ((threadFlag >>  6) & 1) ? traits::identity() : threadScan1[1]);
            tempData.w =
                traits::op(((threadFlag >> 15) & 1) ? traits::identity() : temp,
                       ((threadFlag >>  7) & 1) ? traits::identity() : threadScan1[2]);
        }
    }
    else
    {
        tempData.x = 
            traits::op(((threadFlag >> 12) & 1) ? traits::identity() : temp, threadScan1[0]);    
        tempData.y =
            traits::op(((threadFlag >> 13) & 1) ? traits::identity() : temp, threadScan1[1]);    
        tempData.z =
            traits::op(((threadFlag >> 14) & 1) ? traits::identity() : temp, threadScan1[2]);    
        tempData.w =
            traits::op(((threadFlag >> 15) & 1) ? traits::identity() : temp, threadScan1[3]);    
    }

    // write results to global memory
    if (isLastBlock && !traits::isFullBlock())
    {
        unsigned int i = biDev * 4;
        if (i < numElements) {d_odata[i] = tempData.x;}
        if ((i+1) < numElements) {d_odata[i+1] = tempData.y;}
        if ((i+2) < numElements) {d_odata[i+2] = tempData.z;}
        if ((i+3) < numElements) {d_odata[i+3] = tempData.w;}

    }
    else
    {
        oData[biDev] = tempData;
    }
}

template <class T, class traits, unsigned int blockSize>
__device__ T
reduceCTA(volatile T *s_data)
{
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    T t = s_data[tid];

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { s_data[tid] = t = traits::op(t, s_data[tid + 256]); } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { s_data[tid] = t = traits::op(t, s_data[tid + 128]); } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { s_data[tid] = t = traits::op(t, s_data[tid +  64]); } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { s_data[tid] = t = traits::op(t, s_data[tid + 32]); __EMUSYNC; }
        if (blockSize >=  32) { s_data[tid] = t = traits::op(t, s_data[tid + 16]); __EMUSYNC; }
        if (blockSize >=  16) { s_data[tid] = t = traits::op(t, s_data[tid +  8]); __EMUSYNC; }
        if (blockSize >=   8) { s_data[tid] = t = traits::op(t, s_data[tid +  4]); __EMUSYNC; }
        if (blockSize >=   4) { s_data[tid] = t = traits::op(t, s_data[tid +  2]); __EMUSYNC; }
        if (blockSize >=   2) { s_data[tid] = t = traits::op(t, s_data[tid +  1]); __EMUSYNC; }
    }
    
    // write result for this block to global mem 
    return s_data[0];
}

template<class T, class traits, bool isExclusive, unsigned int maxlevel>
__device__ void warpSegScan(T val,
                            unsigned int flag,
                            volatile T *s_data,
                            volatile unsigned int *s_flags,
                            T& oVal,
                            unsigned int& oFlag)
{
    int idx;
    if (traits::isBackward())
    {
        idx = 2 * threadIdx.x - (threadIdx.x & (WARP_SIZE-1)) + WARP_SIZE;
    }
    else
    {
        idx = 2 * threadIdx.x - (threadIdx.x & (WARP_SIZE-1));
    }

    s_data[idx] = traits::identity(); s_flags[idx] = 0; __EMUSYNC;

    if (traits::isBackward())
    {
        idx -= WARP_SIZE;
    }
    else
    {
        idx += WARP_SIZE;
    }

    T t = s_data[idx] = val; unsigned int f = s_flags[idx] = flag;

    __EMUSYNC;

#ifdef __DEVICE_EMULATION__
    //T t; unsigned int f;
    if (traits::isBackward())
    {
        t = s_data[idx +  1]; f = s_flags[idx +  1]; 
    }
    else
    {
        t = s_data[idx -  1]; f = s_flags[idx -  1];
    }
    
    __EMUSYNC; 
    s_data[idx] = s_flags[idx] ? s_data[idx] : traits::op(s_data[idx],t); 
    s_flags[idx] = f | s_flags[idx];  __EMUSYNC;

    if (traits::isBackward())
    {
        t = s_data[idx +  2]; f = s_flags[idx +  2];
    }
    else
    {
         t = s_data[idx -  2]; f = s_flags[idx -  2]; 
    }
    
    __EMUSYNC; 
    s_data[idx] = s_flags[idx] ? s_data[idx] : traits::op(s_data[idx],t); 
    s_flags[idx] = f | s_flags[idx]; __EMUSYNC;
    
    if (traits::isBackward())
    {
        t = s_data[idx +  4]; f = s_flags[idx +  4];
    }
    else
    {
        t = s_data[idx -  4]; f = s_flags[idx -  4];
    }
    
    __EMUSYNC; 
    s_data[idx] = s_flags[idx] ? s_data[idx] : traits::op(s_data[idx],t); 
    s_flags[idx] = f | s_flags[idx]; __EMUSYNC;

    if (traits::isBackward())
    {
        t = s_data[idx +  8]; f = s_flags[idx +  8];
    }
    else
    {
        t = s_data[idx -  8]; f = s_flags[idx -  8]; 
    }
    
    __EMUSYNC; 
    s_data[idx] = s_flags[idx] ? s_data[idx] : traits::op(s_data[idx],t); 
    s_flags[idx] = f | s_flags[idx]; __EMUSYNC;

    if (traits::isBackward())
    {
        t = s_data[idx + 16]; f = s_flags[idx + 16];
    }
    else
    {
        t = s_data[idx - 16]; f = s_flags[idx - 16]; 
    }
    
    __EMUSYNC; 
    s_data[idx] = s_flags[idx] ? s_data[idx] : traits::op(s_data[idx],t); 
    s_flags[idx] = f | s_flags[idx]; __EMUSYNC;

#else
    if (0 <= maxlevel)
    {
        if (traits::isBackward())
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx + 1] , t);
            s_flags[idx] = f = s_flags[idx + 1] | f;
        }
        else
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx - 1] , t);
            s_flags[idx] = f = s_flags[idx - 1] | f;
        }
    }
    if (1 <= maxlevel)
    {
        if (traits::isBackward())
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx + 2] , t);
            s_flags[idx] = f = s_flags[idx + 2] | f;
        }
        else
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx - 2] , t);
            s_flags[idx] = f = s_flags[idx - 2] | f;
        }
    }
    if (2 <= maxlevel)
    {
        if (traits::isBackward())
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx + 4] , t);
            s_flags[idx] = f = s_flags[idx + 4] | f;
        }
        else
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx - 4] , t);
            s_flags[idx] = f = s_flags[idx - 4] | f;
        }
    }
    if (3 <= maxlevel)
    {
        if (traits::isBackward())
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx + 8] , t);
            s_flags[idx] = f = s_flags[idx + 8] | f;
        }
        else
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx - 8] , t);
            s_flags[idx] = f = s_flags[idx - 8] | f;
        }
    }
    if (4 <= maxlevel)
    {
        if (traits::isBackward())
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx + 16] , t);
            s_flags[idx] = f = s_flags[idx + 16] | f;
        }
        else
        {
            s_data[idx]  = t = f ? t : traits::op(s_data[idx - 16] , t);
            s_flags[idx] = f = s_flags[idx - 16] | f;
        }
    }
#endif

    if( isExclusive ) 
        if (traits::isBackward())
            oVal = (!flag) ? s_data[idx+1] : traits::identity();
        else
            oVal = (!flag) ? s_data[idx-1] : traits::identity();
    else
        oVal =  t;

    oFlag = f;
}


template<class T, class traits>
__device__ void segmentedScanWarps(T val1,
                                   unsigned int flag1,
                                   T val2,
                                   unsigned int flag2,
                                   T *s_data, 
                                   unsigned int *s_flags)
{
    const unsigned int idx = threadIdx.x;

    // Phase 1: Intra-warp prefix sums

    // Seg scan for (0 ... blockDim.x - 1)
    T oVal1; unsigned int oFlag1;
    warpSegScan<T, traits, false, 4>(val1, flag1, s_data, s_flags,
                                     oVal1, oFlag1);
    __syncthreads();

    // Seg scan for (blockDim.x ... 2*blockDim.x - 1)
    T oVal2; unsigned int oFlag2;
    warpSegScan<T, traits, false, 4>(val2, flag2, s_data, s_flags, 
                                     oVal2, oFlag2);
    __syncthreads(); // FIXME - this is needed why?
        
    // Phase 2: Sum across warps of the CTA

    const unsigned int lane   = idx&(WARP_SIZE-1);
    const unsigned int warpid = idx >> LOG_WARP_SIZE;
    const unsigned int warpid2 = (idx + blockDim.x) >> LOG_WARP_SIZE;

    //  - write per-warp partial sums
    if (traits::isBackward())
    {
        const unsigned int num_warps = ((blockDim.x << 1) >> LOG_WARP_SIZE);
        const unsigned int offset = blockDim.x - num_warps;

        // The effect of adding the offset is to shift the input values of the second
        // level seg-scan to the last warp. This is needed because the data movement
        // in backward segmented scan in right to left (as opposed to left to right in 
        // forward segmented scan). This implies that spurious values in other warps
        // will not affect the values that we need in the last warp when we do the 
        // second level segmented scan. This would not have been a concern if the
        // compiler bug noted below is fixed. In that case we could have done a 
        // segmented scan on a single warp instead of a whole CTA
        if( lane == 0 )  
        {
            s_data[warpid + offset] = oVal1; 
            s_data[warpid2 + offset] = oVal2;

            s_flags[warpid + offset] = oFlag1;
            s_flags[warpid2 + offset] = oFlag2;
        }
    }
    else
    {
        if( lane == (WARP_SIZE-1) )  
        {
            s_data[warpid] = oVal1; 
            s_data[warpid2] = oVal2;

            s_flags[warpid] = oFlag1;
            s_flags[warpid2] = oFlag2;
        }
    }
    __syncthreads();

    T oVal3; unsigned int oFlag3;

    T tdata = s_data[idx];
    T tflag = s_flags[idx];
    __syncthreads();

    //  - use 1 warp for prefix sum over them
    // MJH: This optimization saves very little time in practice, and it
    // breaks backward segscans for some reason, so commenting it out.
    //#ifndef __DEVICE_EMULATION__
    //    if ( warpid==0 )   
    //#endif
    {
        warpSegScan<T, traits, false, (LOG_CTA_SIZE-LOG_WARP_SIZE+1)>
            (tdata, tflag, s_data, s_flags, oVal3, oFlag3);
    }
    __syncthreads(); // This looks unnecessary but won't work without it

    s_data[idx] = oVal3;
    s_flags[idx] = oFlag3;
    __syncthreads();

     //  - add the results back into each thread
    if (traits::isBackward())
    {
        // FIXME - this shouldn't need to be defined twice
        const unsigned int num_warps = ((blockDim.x << 1) >> LOG_WARP_SIZE);
        const unsigned int offset = blockDim.x - num_warps;

        oVal1 = oFlag1 ? oVal1 : traits::op(s_data[offset+warpid+1], oVal1);

        if (warpid2 < (num_warps-1)) oVal2 = oFlag2 ? oVal2 : traits::op(s_data[offset+warpid2+1], oVal2);
    }
    else
    {
        if (warpid > 0) oVal1 = oFlag1 ? oVal1 : traits::op(s_data[warpid-1], oVal1);

        oVal2 = oFlag2 ? oVal2 : traits::op(s_data[warpid2-1], oVal2);
    }
    __syncthreads(); // This looks unnecessary

     //  - and we're done
     s_data[idx] = oVal1;
     s_data[idx + blockDim.x] = oVal2;
     
     __syncthreads(); // make sure the caller sees all our s_data[] writes
}


/**
* @brief CTA-level segmented scan routine; 
* 
* Performs segmented scan on \a s_data in shared memory in each thread block 
* with head flags in \a s_flags (\a s_tflags is a read-write copy of the head 
* flags which are modified).
*
* This function is the main CTA-level segmented scan function.  It may be called 
* by other CUDA __global__ or __device__ functions.
* \note This code is intended to be run on a CTA of 128 threads.  Other sizes are
* untested.
* 
* @param[in] s_data Array to be scanned in shared memory
* @param[in] s_flags Read-only version of flags in shared memory
* @param[in] s_indices Temporary read-write indices array
* @param[out] d_blockSums Array of per-block sums
* @param[out] d_blockFlags Array of per-block OR-reduction of flags
* @param[out] d_blockIndices Array of per-block min-reduction of indices
*/
template<class T, class traits>
__device__
void segmentedScanCTA(T            *s_data, 
                      unsigned int *s_flags,
                      unsigned int *s_indices,
                      T            *d_blockSums = 0, 
                      unsigned int *d_blockFlags = 0,
                      unsigned int *d_blockIndices = 0)
{
    T val = s_data[threadIdx.x];
    T val2 = s_data[threadIdx.x + blockDim.x];
    unsigned int flag = s_flags[threadIdx.x];
    unsigned int flag2 = s_flags[threadIdx.x + blockDim.x];

    unsigned int cta_is_closed = s_flags[0];

    __syncthreads();

    segmentedScanWarps<T, traits>(val, flag, val2, flag2, 
                                  s_data, s_flags);

    if (traits::isBackward())
    {
        if (traits::writeSums() && (threadIdx.x == 0))
        {
            d_blockSums[blockIdx.x] = s_data[0];

            const unsigned int num_warps = ((blockDim.x << 1) >> LOG_WARP_SIZE);
            const unsigned int offset = blockDim.x - num_warps;
            d_blockFlags[blockIdx.x] = (s_flags[offset+0] != 0);
        }
    }
    else
    {
        if (traits::writeSums() && (threadIdx.x == blockDim.x - 1))
        {
            d_blockSums[blockIdx.x] = s_data[threadIdx.x + blockDim.x];
            d_blockFlags[blockIdx.x] = cta_is_closed || (s_flags[(1 << (LOG_CTA_SIZE-LOG_WARP_SIZE+1))-1] != 0);
        }
    }

    unsigned int mIndex;

    if (traits::writeSums())
    {
        if (traits::isBackward())
        {
            mIndex = 
                reduceCTA<unsigned int, ScanTraits<unsigned int, CUDPP_MAX, false, false, false, false, true>,
                      (2 * CTA_SIZE)>(s_indices);
        }
        else
        {
            mIndex =
                reduceCTA<unsigned int, ScanTraits<unsigned int, CUDPP_MIN, false, false, false, false, true>,
                      (2 * CTA_SIZE)>(s_indices);
        }
    }

    if (traits::writeSums() && (threadIdx.x == 0))
    {
        d_blockIndices[blockIdx.x] = mIndex;
    }
}

/** @} */ // end segmented scan functions
/** @} */ // end cudpp_cta
