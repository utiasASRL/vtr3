#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision: 5633 $
// $Date: 2009-07-01 15:02:51 +1000 (Wed, 01 Jul 2009) $
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * scan_app.cu
 *
 * @brief CUDPP application-level scan routines
 */

/** \defgroup cudpp_app CUDPP Application-Level API
  * The CUDPP Application-Level API contains functions
  * that run on the host CPU and invoke GPU routines in 
  * the CUDPP \link cudpp_kernel Kernel-Level API\endlink. 
  * Application-Level API functions are used by
  * CUDPP \link publicInterface Public Interface\endlink
  * functions to implement CUDPP's core functionality.
  * @{
  */

/** @name Scan Functions
 * @{
 */

#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_plan.h"
#include "kernel/scan_kernel.cu"
#include "kernel/vector_kernel.cu"


#include <cutil.h>
#include <cstdlib>
#include <cstdio>
#include <assert.h>

/** @brief Perform recursive scan on arbitrary size arrays
  *
  * This is the CPU-side workhorse function of the scan engine.  This function
  * invokes the CUDA kernels which perform the scan on individual blocks. 
  *
  * Scans of large arrays must be split (possibly recursively) into a hierarchy of block scans,
  * where each block is scanned by a single CUDA thread block.  At each recursive level of the
  * scanArrayRecursive first invokes a kernel to scan all blocks of that level, and if the level
  * has more than one block, it calls itself recursively.  On returning from each recursive level,
  * the total sum of each block from the level below is added to all elements of the corresponding
  * block in this level.  See "Parallel Prefix Sum (Scan) in CUDA" for more information (see
  * \ref references ).
  * 
  * Template parameter \a T is the datatype; \a isBackward specifies backward or forward scan; 
  * \a isExclusive specifies exclusive or inclusive scan, and \a op specifies the binary associative
  * operator to be used.
  *
  * @param[out] d_out       The output array for the scan results
  * @param[in]  d_in        The input array to be scanned
  * @param[out] d_blockSums Array of arrays of per-block sums (one array per recursive level, allocated
  *                         by allocScanStorage())
  * @param[in]  numElements The number of elements in the array to scan
  * @param[in]  numRows The number of rows in the array to scan
  * @param[in]  rowPitches  Array of row pitches (one array per recursive level, allocated by 
  *                         allocScanStorage())
  * @param[in]  level       The current recursive level of the scan
  */
template <class T, bool isBackward, bool isExclusive, CUDPPOperator op>
void scanArrayRecursive(T                   *d_out, 
                        const T             *d_in, 
                        T                   **d_blockSums,
                        size_t              numElements,
                        size_t              numRows,
                        const size_t        *rowPitches,
                        int                 level)
{
    unsigned int numBlocks = 
        max(1, (unsigned int)ceil((double)numElements / ((double)SCAN_ELTS_PER_THREAD * CTA_SIZE)));

    unsigned int sharedEltsPerBlock = CTA_SIZE * 2;
      
    unsigned int sharedMemSize = sizeof(T) * sharedEltsPerBlock;

    // divide pitch by four since scan's load/store addresses are for vec4 elements
    unsigned int rowPitch = 1;
    unsigned int blockSumRowPitch = 1;

    if (numRows > 1)
    {
        rowPitch         = rowPitches[level] / 4; 
        blockSumRowPitch = (numBlocks > 1) ? rowPitches[level+1] / 4 : 0;
    }

    bool fullBlock = (numElements == numBlocks * SCAN_ELTS_PER_THREAD * CTA_SIZE);

    // setup execution parameters
    dim3  grid(numBlocks, numRows, 1); 
    dim3  threads(CTA_SIZE, 1, 1);

    // make sure there are no CUDA errors before we start
    CUT_CHECK_ERROR("scanArray before kernels");

    unsigned int traitsCode = 0;
    if (numBlocks > 1) traitsCode |= 1;
    if (numRows > 1)   traitsCode |= 2;
    if (fullBlock)     traitsCode |= 4;

    switch (traitsCode)
    {
    case 0: // single block, single row, non-full block
        scan4<T, ScanTraits<T, op, isBackward, isExclusive, false, false, false> >
               <<< grid, threads, sharedMemSize >>>
               (d_out, d_in, 0, numElements, rowPitch, blockSumRowPitch);
        break;
    case 1: // multiblock, single row, non-full block
        scan4< T, ScanTraits<T, op, isBackward, isExclusive, false, true, false> >
               <<< grid, threads, sharedMemSize >>>
               (d_out, d_in, d_blockSums[level], numElements, rowPitch, blockSumRowPitch);
        break;
    case 2: // single block, multirow, non-full block
        scan4<T, ScanTraits<T, op, isBackward, isExclusive, true, false, false> >
                <<< grid, threads, sharedMemSize >>>
                (d_out, d_in, 0, numElements, rowPitch, blockSumRowPitch);
        break;
    case 3: // multiblock, multirow, non-full block
        scan4<T, ScanTraits<T, op, isBackward, isExclusive, true, true, false> >
                <<< grid, threads, sharedMemSize >>>
                (d_out, d_in, d_blockSums[level], numElements, rowPitch, blockSumRowPitch);
        break;
    case 4: // single block, single row, full block
        scan4<T, ScanTraits<T, op, isBackward, isExclusive, false, false, true> >
               <<< grid, threads, sharedMemSize >>>
               (d_out, d_in, 0, numElements, rowPitch, blockSumRowPitch);
        break;
    case 5: // multiblock, single row, full block
        scan4< T, ScanTraits<T, op, isBackward, isExclusive, false, true, true> >
               <<< grid, threads, sharedMemSize >>>
               (d_out, d_in, d_blockSums[level], numElements, rowPitch, blockSumRowPitch);
        break;
    case 6: // single block, multirow, full block
        scan4<T, ScanTraits<T, op, isBackward, isExclusive, true, false, true> >
                <<< grid, threads, sharedMemSize >>>
                (d_out, d_in, 0, numElements, rowPitch, blockSumRowPitch);
        break;
    case 7: // multiblock, multirow, full block
        scan4<T, ScanTraits<T, op, isBackward, isExclusive, true, true, true> >
                <<< grid, threads, sharedMemSize >>>
                (d_out, d_in, d_blockSums[level], numElements, rowPitch, blockSumRowPitch);
        break;
    }

    CUT_CHECK_ERROR("prescan");

    if (numBlocks > 1)
    {
        // After scanning all the sub-blocks, we are mostly done. But
        // now we need to take all of the last values of the
        // sub-blocks and scan those. This will give us a new value
        // that must be sdded to each block to get the final results.

        scanArrayRecursive<T, isBackward, true, op>
            ((T*)d_blockSums[level], (const T*)d_blockSums[level],
             (T**)d_blockSums, numBlocks, numRows, rowPitches, level + 1); // recursive (CPU) call
        
        vectorAddUniform4<T, op, SCAN_ELTS_PER_THREAD>
            <<< grid, threads >>>(d_out, 
                                  (T*)d_blockSums[level], 
                                  numElements,
                                  rowPitch*4,
                                  blockSumRowPitch*4,
                                  0, 0);
        CUT_CHECK_ERROR("vectorAddUniform");
    }
}

// global
    
#ifdef __cplusplus
extern "C" 
{
#endif

/** @brief Allocate intermediate arrays used by scan.
  *
  * Scans of large arrays must be split (possibly recursively) into a hierarchy 
  * of block scans, where each block is scanned by a single CUDA thread block.  
  * At each recursive level of the scan, we need an array in which to store the 
  * total sums of all blocks in that level.  This function computes the amount 
  * of storage needed and allocates it.
  *
  * @param plan Pointer to CUDPPScanPlan object containing options and number 
  *             of elements, which is used to compute storage requirements, and
  *             within which intermediate storage is allocated.
  */
void allocScanStorage(CUDPPScanPlan *plan)
{
    //assert(config->_numEltsAllocated == 0); // shouldn't be called 

    plan->m_numEltsAllocated = plan->m_numElements;

    size_t numElts = plan->m_numElements;
    
    size_t level = 0;

    do
    {       
        size_t numBlocks = 
            max(1, (unsigned int)ceil((double)numElts / ((double)SCAN_ELTS_PER_THREAD * CTA_SIZE)));
        if (numBlocks > 1)
        {
            level++;
        }
        numElts = numBlocks;
    } while (numElts > 1);

    size_t elementSize = 0;

    switch(plan->m_config.datatype)
    {
    case CUDPP_INT:
        plan->m_blockSums = (void**) malloc(level * sizeof(int*));
        elementSize = sizeof(int);
        break;
    case CUDPP_UINT:
        plan->m_blockSums = (void**) malloc(level * sizeof(unsigned int*));
        elementSize = sizeof(unsigned int);
        break;
    case CUDPP_FLOAT:
        plan->m_blockSums = (void**) malloc(level * sizeof(float*));
        elementSize = sizeof(float);
        break;
    default:
        break;
    }

    plan->m_numLevelsAllocated = level;
    numElts = plan->m_numElements;
    size_t numRows = plan->m_numRows;
    plan->m_numRowsAllocated = numRows;
    plan->m_rowPitches = 0;

    if (numRows > 1)
    {
        plan->m_rowPitches = (size_t*) malloc((level + 1) * sizeof(size_t));
        plan->m_rowPitches[0] = plan->m_rowPitch;
    }

    level = 0;

    do
    {       
        size_t numBlocks = 
            max(1, (unsigned int)ceil((double)numElts / ((double)SCAN_ELTS_PER_THREAD * CTA_SIZE)));
        if (numBlocks > 1) 
        {
            // Use hipMallocPitch for multi-row block sums to ensure alignment
            if (numRows > 1)
            {
                size_t dpitch;
                CUDA_SAFE_CALL( hipMallocPitch((void**) &(plan->m_blockSums[level]), 
                                                &dpitch,
                                                numBlocks * elementSize, 
                                                numRows));
                plan->m_rowPitches[level+1] = dpitch / elementSize;
                level++;
            }
            else
            {
                CUDA_SAFE_CALL(hipMalloc((void**) &(plan->m_blockSums[level++]),  
                                          numBlocks * elementSize));
            }
        }
        numElts = numBlocks;
    } while (numElts > 1);

    CUT_CHECK_ERROR("allocScanStorage");
}

/** @brief Deallocate intermediate block sums arrays in a CUDPPScanPlan object.
  *
  * These arrays must have been allocated by allocScanStorage(), which is called
  * by the constructor of cudppScanPlan().  
  *
  * @param plan Pointer to CUDPPScanPlan object initialized by allocScanStorage().
  */
void freeScanStorage(CUDPPScanPlan *plan)
{
    for (unsigned int i = 0; i < plan->m_numLevelsAllocated; i++)
    {
        hipFree(plan->m_blockSums[i]);
    }

    CUT_CHECK_ERROR("freeScanStorage");

    free((void**)plan->m_blockSums);
    if (plan->m_numRows > 1)
        free((void*)plan->m_rowPitches);

    plan->m_blockSums = 0;
    plan->m_numEltsAllocated = 0;
    plan->m_numLevelsAllocated = 0;
}


/** @brief Dispatch function to perform a scan (prefix sum) on an
  * array with the specified configuration.
  *
  * This is the dispatch routine which calls scanArrayRecursive() with 
  * appropriate template parameters and arguments to achieve the scan as 
  * specified in \a plan. 
  * 
  * @param[out] d_out    The output array of scan results
  * @param[in]  d_in     The input array
  * @param[in]  numElements The number of elements to scan
  * @param[in]  numRows     The number of rows to scan in parallel
  * @param[in]  plan     Pointer to CUDPPScanPlan object containing scan options
  *                      and intermediate storage
  */
void cudppScanDispatch(void                *d_out, 
                       const void          *d_in, 
                       size_t              numElements,
                       size_t              numRows,
                       const CUDPPScanPlan *plan)
{    
    if (CUDPP_OPTION_EXCLUSIVE & plan->m_config.options)
    {
        if (CUDPP_OPTION_BACKWARD & plan->m_config.options)
        {
            switch (plan->m_config.datatype)
            {
            case CUDPP_INT:

                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<int, true, true, CUDPP_ADD>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<int, true, true, CUDPP_MULTIPLY>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<int, true, true, CUDPP_MAX>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<int, true, true, CUDPP_MIN>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }
              
                break;

            case CUDPP_UINT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:                 
                    scanArrayRecursive<unsigned int, true, true, CUDPP_ADD>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:                 
                    scanArrayRecursive<unsigned int, true, true, CUDPP_MULTIPLY>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<unsigned int, true, true, CUDPP_MAX>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<unsigned int, true, true, CUDPP_MIN>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }

                break;

            case CUDPP_FLOAT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<float, true, true,  CUDPP_ADD>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<float, true, true,  CUDPP_MULTIPLY>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<float, true, true, CUDPP_MAX>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<float, true, true, CUDPP_MIN>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }
                break; 

            default:
                break; 
            }
        }
        else
        {
            switch (plan->m_config.datatype)
            {
            case CUDPP_INT:

                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<int, false, true, CUDPP_ADD>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<int, false, true, CUDPP_MULTIPLY>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<int, false, true, CUDPP_MAX>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<int, false, true, CUDPP_MIN>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }

                break;
                    
            case CUDPP_UINT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:                 
                    scanArrayRecursive<unsigned int, false, true, CUDPP_ADD>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:                 
                    scanArrayRecursive<unsigned int, false, true, CUDPP_MULTIPLY>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<unsigned int, false, true, CUDPP_MAX>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<unsigned int, false, true, CUDPP_MIN>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                            
                }
        
                break;       
            
            case CUDPP_FLOAT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<float, false, true, CUDPP_ADD>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<float, false, true, CUDPP_MULTIPLY>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<float, false, true, CUDPP_MAX>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<float, false, true, CUDPP_MIN>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }            
                break;

            default:
                break; 
            }
        }
    }
    else
    {
        if (CUDPP_OPTION_BACKWARD & plan->m_config.options)
        {
            switch (plan->m_config.datatype)
            {
            case CUDPP_INT:

                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<int, true, false, CUDPP_ADD>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<int, true, false, CUDPP_MULTIPLY>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<int, true, false, CUDPP_MAX>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<int, true, false, CUDPP_MIN>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }
              
                break;

            case CUDPP_UINT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:                 
                    scanArrayRecursive<unsigned int, true, false, CUDPP_ADD>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:                 
                    scanArrayRecursive<unsigned int, true, false, CUDPP_MULTIPLY>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<unsigned int, true, false, CUDPP_MAX>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<unsigned int, true, false, CUDPP_MIN>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }

                break;

            case CUDPP_FLOAT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<float, true, false, CUDPP_ADD>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<float, true, false, CUDPP_MULTIPLY>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<float, true, false, CUDPP_MAX>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<float, true, false, CUDPP_MIN>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }
                break; 

            default:
                break; 
            }
        }
        else
        {
            switch (plan->m_config.datatype)
            {
            case CUDPP_INT:

                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<int, false, false, CUDPP_ADD>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<int, false, false, CUDPP_MULTIPLY>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<int, false, false, CUDPP_MAX>
                        ((int*)d_out, (const int*)d_in, 
                         (int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<int, false, false, CUDPP_MIN>
                        ((int*)d_out, (const int*)d_in, 
                        (int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }

                break;
                    
            case CUDPP_UINT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:                 
                    scanArrayRecursive<unsigned int, false, false, CUDPP_ADD>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:                 
                    scanArrayRecursive<unsigned int, false, false, CUDPP_MULTIPLY>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<unsigned int, false, false, CUDPP_MAX>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                         (unsigned int**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<unsigned int, false, false, CUDPP_MIN>
                        ((unsigned int*)d_out, (const unsigned int*)d_in, 
                        (unsigned int**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                            
                }
        
                break;       
            
            case CUDPP_FLOAT:
                switch(plan->m_config.op)
                {
                case CUDPP_ADD:
                    scanArrayRecursive<float, false, false, CUDPP_ADD>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MULTIPLY:
                    scanArrayRecursive<float, false, false, CUDPP_MULTIPLY>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MAX:
                    scanArrayRecursive<float, false, false, CUDPP_MAX>
                        ((float*)d_out, (const float*)d_in, 
                         (float**)plan->m_blockSums, 
                         numElements, numRows, plan->m_rowPitches, 0);
                    break;
                case CUDPP_MIN:
                    scanArrayRecursive<float, false, false, CUDPP_MIN>
                        ((float*)d_out, (const float*)d_in, 
                        (float**)plan->m_blockSums, 
                        numElements, numRows, plan->m_rowPitches, 0);
                    break;
                default:
                    break;
                }            
                break;

            default:
                break; 
            }
        }  
    }
}

#ifdef __cplusplus
}
#endif

/** @} */ // end scan functions
/** @} */ // end cudpp_app
