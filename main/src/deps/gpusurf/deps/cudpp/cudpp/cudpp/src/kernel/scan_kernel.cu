#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
//  $Revision: 5633 $
//  $Date: 2009-07-01 15:02:51 +1000 (Wed, 01 Jul 2009) $
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * scan_kernel.cu
 *
 * @brief CUDPP kernel-level scan routines
 */

/** \defgroup cudpp_kernel CUDPP Kernel-Level API
  * The CUDPP Kernel-Level API contains functions that run on the GPU 
  * device across a grid of Cooperative Thread Array (CTA, aka Thread
  * Block).  These kernels are declared \c __global__ so that they 
  * must be invoked from host (CPU) code.  They generally invoke GPU 
  * \c __device__ routines in the CUDPP \link cudpp_cta CTA-Level API\endlink. 
  * Kernel-Level API functions are used by CUDPP 
  * \link cudpp_app Application-Level\endlink functions to implement their 
  * functionality.
  * @{
  */

/** @name Scan Functions
* @{
*/

#include <cudpp_globals.h>
#include "cta/scan_cta.cu"
#include "sharedmem.h"

/**
  * @brief Main scan kernel
  *
  * This __global__ device function performs one level of a multiblock scan on 
  * an arbitrary-dimensioned array in \a d_in, returning the result in \a d_out 
  * (which may point to the same array).  The same function may be used for
  * single or multi-row scans.  To perform a multirow scan, pass the width of 
  * each row of the input row (in elements) in \a dataRowPitch, and the width of 
  * the rows of \a d_blockSums (in elements) in \a blockSumRowPitch, and invoke
  * with a thread block grid with height greater than 1.  
  * 
  * This function peforms one level of a recursive, multiblock scan.  At the 
  * app level, this function is called by cudppScan and cudppMultiScan and used 
  * in combination with vectorAddUniform4() to produce a complete scan.
  *
  * Template parameter \a T is the datatype of the array to be scanned. 
  * Template parameter \a traits is the ScanTraits struct containing 
  * compile-time options for the scan, such as whether it is forward or 
  * backward, exclusive or inclusive, multi- or single-row, etc.
  * 
  * @param[out] d_out The output (scanned) array
  * @param[in]  d_in The input array to be scanned
  * @param[out] d_blockSums The array of per-block sums
  * @param[in]  numElements The number of elements to scan
  * @param[in]  dataRowPitch The width of each row of \a d_in in elements 
  * (for multi-row scans)
  * @param[in]  blockSumRowPitch The with of each row of \a d_blockSums in elements
  * (for multi-row scans)
  */
template<class T, class traits> 
__global__ void scan4(T            *d_out, 
                      const T      *d_in, 
                      T            *d_blockSums, 
                      int          numElements, 
                      unsigned int dataRowPitch,
                      unsigned int blockSumRowPitch)
{
    SharedMemory<T> smem;
    T* temp = smem.getPointer();

    int devOffset, ai, bi, aiDev, biDev;
    T threadScan0[4], threadScan1[4];

    unsigned int blockN = numElements;
    unsigned int blockSumIndex = blockIdx.x;

    if (traits::isMultiRow())
    {
        //int width = __mul24(gridDim.x, blockDim.x) << 1;
        int yIndex     = __umul24(blockDim.y, blockIdx.y) + threadIdx.y;
        devOffset      = __umul24(dataRowPitch, yIndex);
        blockN        += (devOffset << 2);
        devOffset     += __umul24(blockIdx.x, blockDim.x << 1);
        blockSumIndex += __umul24(blockSumRowPitch << 2, yIndex) ;
    }
    else
    {
        devOffset = __umul24(blockIdx.x, (blockDim.x << 1));
    }
    
    // load data into shared memory
    loadSharedChunkFromMem4<T, traits>
        (temp, threadScan0, threadScan1, d_in,
         blockN, devOffset, ai, bi, aiDev, biDev);

    scanCTA<T, traits>(temp, d_blockSums, blockSumIndex);
    
    // write results to device memory
    storeSharedChunkToMem4<T, traits>
        (d_out, threadScan0, threadScan1, temp, 
         blockN, devOffset, ai, bi, aiDev, biDev);

}

/** @} */ // end scan functions
/** @} */ // end cudpp_kernel
