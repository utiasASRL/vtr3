#include "hip/hip_runtime.h"
// ------------------------------------------------------------- 
//  cuDPP -- CUDA Data Parallel Primitives library
//  -------------------------------------------------------------
//  $Revision: 5633 $
//  $Date: 2009-07-01 15:02:51 +1000 (Wed, 01 Jul 2009) $
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * scan_cta.cu
 *
 * @brief CUDPP CTA-level scan routines
 */

/** \defgroup cudpp_cta CUDPP CTA-Level API
  * The CUDPP CTA-Level API contains functions that run on the GPU 
  * device.  These are CUDA \c __device__ functions that are called
  * from within other CUDA device functions (typically 
  * \link cudpp_kernel CUDPP Kernel-Level API\endlink functions).
  * They are called CTA-level functions because they typically process
  * s_data "owned" by each CTA within shared memory, and are agnostic of
  * any other CTAs that may be running (or how many CTAs are running),
  * other than to compute appropriate global memory addresses.
  * @{
  */

/** @name Scan Functions
* @{
*/

#include <cudpp_globals.h>
#include <cudpp_util.h>
#include <math.h>
#include <cudpp.h>

/**
 * @brief Macro to insert necessary __syncthreads() in device emulation mode
 */
#ifdef __DEVICE_EMULATION__
#define __EMUSYNC __syncthreads()
#else
#define __EMUSYNC
#endif

/** 
  * @brief Template class containing compile-time parameters to the scan functions
  *
  * ScanTraits is passed as a template parameter to all scan functions.  By 
  * using these compile-time functions we can enable generic code while 
  * maintaining the highest performance.  This is crucial for the performance 
  * of low-level workhorse algorithms like scan.
  *
  * @param T The datatype of the scan
  * @param oper The ::CUDPPOperator to use for the scan (add, max, etc.)
  * @param multiRow True if this is a multi-row scan
  * @param unroll True if scan inner loops should be unrolled
  * @param sums True if each block should write it's sum to the d_blockSums array (false for single-block scans)
  * @param backward True if this is a backward scan
  * @param fullBlock True if all blocks in this scan are full (CTA_SIZE * SCAN_ELEMENTS_PER_THREAD elements)
  * @param exclusive True for exclusive scans, false for inclusive scans
  */
template <class T, CUDPPOperator oper, bool backward, bool exclusive,
          bool multiRow, bool sums, bool fullBlock>
class ScanTraits
{
public:
    
    //! Returns true if this is a backward scan
    static __device__ bool isBackward()    { return backward; };
    //! Returns true if this is an exclusive scan
    static __device__ bool isExclusive()  { return exclusive; };
    //! Returns true if this a multi-row scan.
    static __device__ bool isMultiRow()    { return multiRow; };
    //! Returns true if this scan writes the sum of each block to the d_blockSums array (multi-block scans)
    static __device__ bool writeSums()     { return sums; };
    //! Returns true if this is a full scan -- all blocks process CTA_SIZE * SCAN_ELEMENTS_PER_THREAD elements
    static __device__ bool isFullBlock()   { return fullBlock; };
    
        
    //! The operator function used for the scan
    static __device__ T op(const T a, const T b)
    {
        return Operator<T, oper>::op(a, b);
    }  

    //! The identity value used by the scan
    static __device__ T identity() { return Operator<T, oper>::identity(); }
};

//! This is used to insert syncthreads to avoid perf loss caused by 128-bit 
//! load overlap that happens on G80.  This gives about a 15% boost on scans on 
//! G80.
//! @todo Parameterize this in case this perf detail changes on future GPUs.
#define DISALLOW_LOADSTORE_OVERLAP 1

/**
* @brief Handles loading input s_data from global memory to shared memory 
* (vec4 version)
*
* Load a chunk of 8*blockDim.x elements from global memory into a 
* shared memory array.  Each thread loads two T4 elements (where
* T4 is, e.g. int4 or float4), computes the scan of those two vec4s in 
* thread local arrays (in registers), and writes the two total sums of the
* vec4s into shared memory, where they will be cooperatively scanned with 
* the other partial sums by all threads in the CTA.
*
* @param[out] s_out The output (shared) memory array
* @param[out] threadScan0 Intermediate per-thread partial sums array 1
* @param[out] threadScan1 Intermediate per-thread partial sums array 2
* @param[in] d_in The input (device) memory array
* @param[in] numElements The number of elements in the array being scanned
* @param[in] iDataOffset the offset of the input array in global memory for this 
* thread block
* @param[out] ai The shared memory address for the thread's first element 
* (returned for reuse)
* @param[out] bi The shared memory address for the thread's second element 
* (returned for reuse)
* @param[out] aiDev The device memory address for this thread's first element 
* (returned for reuse)
* @param[out] biDev The device memory address for this thread's second element 
* (returned for reuse)
*/
template <class T, class traits> 
__device__ void loadSharedChunkFromMem4(T        *s_out,
                                        T        threadScan0[4],
                                        T        threadScan1[4],
                                        const T  *d_in,
                                        int      numElements, 
                                        int      iDataOffset,
                                        int      &ai, 
                                        int      &bi, 
                                        int      &aiDev, 
                                        int      &biDev)
{
    int thid = threadIdx.x;
    aiDev = iDataOffset + thid;
    biDev = aiDev + blockDim.x;

    // convert to 4-vector
    typename typeToVector<T,4>::Result  tempData;
    typename typeToVector<T,4>::Result* inData = (typename typeToVector<T,4>::Result*)d_in;

    ai = thid;
    bi = thid + blockDim.x;

    // read into tempData;
    if (traits::isBackward())
    {
        int i = aiDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements) 
        {
            tempData       = inData[aiDev];
            threadScan0[3] = tempData.w;               
            threadScan0[2] = traits::op(tempData.z, threadScan0[3]);
            threadScan0[1] = traits::op(tempData.y, threadScan0[2]);
            threadScan0[0] = s_out[ai] 
                           = traits::op(tempData.x, threadScan0[1]);
        }
        else
        {
            threadScan0[3] = traits::identity();
            threadScan0[2] = traits::op(((i+2) < numElements) ? d_in[i+2] : traits::identity(), threadScan0[3]);
            threadScan0[1] = traits::op(((i+1) < numElements) ? d_in[i+1] : traits::identity(), threadScan0[2]);
            threadScan0[0] = s_out[ai] 
                           = traits::op((i     < numElements) ? d_in[i]   : traits::identity(), threadScan0[1]);
        }

#ifdef DISALLOW_LOADSTORE_OVERLAP
        __syncthreads();
#endif

        i = biDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {
            tempData       = inData[biDev];
            threadScan1[3] = tempData.w;
            threadScan1[2] = traits::op(tempData.z, threadScan1[3]);
            threadScan1[1] = traits::op(tempData.y, threadScan1[2]);
            threadScan1[0] = s_out[bi] 
                           = traits::op(tempData.x, threadScan1[1]);
        }
        else
        {
            threadScan1[3] = traits::identity();
            threadScan1[2] = traits::op(((i+2) < numElements) ? d_in[i+2] : traits::identity(), threadScan1[3]);
            threadScan1[1] = traits::op(((i+1) < numElements) ? d_in[i+1] : traits::identity(), threadScan1[2]);
            threadScan1[0] = s_out[bi] 
                           = traits::op((i     < numElements) ? d_in[i]   : traits::identity(), threadScan1[1]);
        }
        __syncthreads();

        // reverse s_data in shared memory
        if (ai < CTA_SIZE)
        {       
            unsigned int leftIdx = ai;
            unsigned int rightIdx = (2 * CTA_SIZE - 1) - ai;
                
            if (leftIdx < rightIdx) 
            {
                T tmp           = s_out[leftIdx];
                s_out[leftIdx]  = s_out[rightIdx];
                s_out[rightIdx] = tmp;
            }
        }
        __syncthreads();
    }
    else
    {
        int i = aiDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {
            tempData       = inData[aiDev];
            threadScan0[0] = tempData.x;           
            threadScan0[1] = traits::op(tempData.y, threadScan0[0]);
            threadScan0[2] = traits::op(tempData.z, threadScan0[1]);
            threadScan0[3] = s_out[ai] 
                           = traits::op(tempData.w, threadScan0[2]);
        }
        else
        {
            threadScan0[0] = (i < numElements) ? d_in[i] : traits::identity();
            threadScan0[1] = traits::op(((i+1) < numElements) ? d_in[i+1] : traits::identity(), threadScan0[0]);
            threadScan0[2] = traits::op(((i+2) < numElements) ? d_in[i+2] : traits::identity(), threadScan0[1]);
            threadScan0[3] = s_out[ai] 
                           = traits::op(((i+3) < numElements) ? d_in[i+3] : traits::identity(), threadScan0[2]);
        }

        
#ifdef DISALLOW_LOADSTORE_OVERLAP
        __syncthreads();
#endif

        i = biDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {
            tempData       = inData[biDev];
            threadScan1[0] = tempData.x;           
            threadScan1[1] = traits::op(tempData.y, threadScan1[0]);
            threadScan1[2] = traits::op(tempData.z, threadScan1[1]);
            threadScan1[3] = s_out[bi] 
                           = traits::op(tempData.w, threadScan1[2]);
        }
        else
        {
            threadScan1[0] = (i < numElements) ? d_in[i] : traits::identity();
            threadScan1[1] = traits::op(((i+1) < numElements) ? d_in[i+1] : traits::identity(), threadScan1[0]);
            threadScan1[2] = traits::op(((i+2) < numElements) ? d_in[i+2] : traits::identity(), threadScan1[1]);
            threadScan1[3] = s_out[bi] 
                           = traits::op(((i+3) < numElements) ? d_in[i+3] : traits::identity(), threadScan1[2]);
        }  
        __syncthreads();
    }
}


/**
* @brief Handles storing result s_data from shared memory to global memory 
* (vec4 version)
*
* Store a chunk of SCAN_ELTS_PER_THREAD*blockDim.x elements from shared memory 
* into a device memory array.  Each thread stores reads two elements from shared
* memory, adds them to the intermediate sums computed in 
* loadSharedChunkFromMem4(), and writes two T4 elements (where
* T4 is, e.g. int4 or float4) to global memory.
*
* @param[out] d_out The output (device) memory array
* @param[in] threadScan0 Intermediate per-thread partial sums array 1
* (contents computed in loadSharedChunkFromMem4())
* @param[in] threadScan1 Intermediate per-thread partial sums array 2
* (contents computed in loadSharedChunkFromMem4())
* @param[in] s_in The input (shared) memory array
* @param[in] numElements The number of elements in the array being scanned
* @param[in] oDataOffset the offset of the output array in global memory 
* for this thread block
* @param[in] ai The shared memory address for the thread's first element 
* (computed in loadSharedChunkFromMem4())
* @param[in] bi The shared memory address for the thread's second element 
* (computed in loadSharedChunkFromMem4())
* @param[in] aiDev The device memory address for this thread's first element 
* (computed in loadSharedChunkFromMem4())
* @param[in] biDev The device memory address for this thread's second element 
* (computed in loadSharedChunkFromMem4())
*/
template <class T, class traits>
__device__ void storeSharedChunkToMem4(T   *d_out,
                                       T   threadScan0[4],
                                       T   threadScan1[4],
                                       T   *s_in,
                                       int numElements, 
                                       int oDataOffset,
                                       int ai, 
                                       int bi, 
                                       int aiDev, 
                                       int biDev)
{
    // Convert to 4-vector
    typename typeToVector<T,4>::Result tempData;
    typename typeToVector<T,4>::Result* outData = (typename typeToVector<T,4>::Result*)d_out;

    // write results to global memory
    if (traits::isBackward())
    {   
        if (ai < CTA_SIZE)
        {

            unsigned int leftIdx = ai;
            unsigned int rightIdx = (2 * CTA_SIZE - 1) - ai;
            
            if (leftIdx < rightIdx) 
            {
                T tmp = s_in[leftIdx];
                s_in[leftIdx] = s_in[rightIdx];
                s_in[rightIdx] = tmp;
            }
        }
        __syncthreads();

        T temp = s_in[ai];

        if (traits::isExclusive())
        {
            tempData.w = temp;
            tempData.z = traits::op(temp, threadScan0[3]);
            tempData.y = traits::op(temp, threadScan0[2]);
            tempData.x = traits::op(temp, threadScan0[1]);
        }
        else
        {
            tempData.w = traits::op(temp, threadScan0[3]);
            tempData.z = traits::op(temp, threadScan0[2]);
            tempData.y = traits::op(temp, threadScan0[1]);
            tempData.x = traits::op(temp, threadScan0[0]);
        }

        int i = aiDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {
            outData[aiDev] = tempData;
        }
        else
        {
            if (i   < numElements) { d_out[i]   = tempData.x;
            if (i+1 < numElements) { d_out[i+1] = tempData.y;
            if (i+2 < numElements) { d_out[i+2] = tempData.z; }}}     
        }

#ifdef DISALLOW_LOADSTORE_OVERLAP
        __syncthreads();
#endif

        temp = s_in[bi];

        if (traits::isExclusive())
        {
            tempData.w = temp;
            tempData.z = traits::op(temp, threadScan1[3]);
            tempData.y = traits::op(temp, threadScan1[2]);
            tempData.x = traits::op(temp, threadScan1[1]);
        }
        else
        {
            tempData.w = traits::op(temp, threadScan1[3]);
            tempData.z = traits::op(temp, threadScan1[2]);
            tempData.y = traits::op(temp, threadScan1[1]);
            tempData.x = traits::op(temp, threadScan1[0]);
        }

        i = biDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {
            outData[biDev] = tempData;
        }
        else
        {
            if (i   < numElements) { d_out[i]   = tempData.x;
            if (i+1 < numElements) { d_out[i+1] = tempData.y;
            if (i+2 < numElements) { d_out[i+2] = tempData.z; }}}     
        }
    }
    else
    {
        T temp;
        temp = s_in[ai]; 

        if (traits::isExclusive())
        {
            tempData.x = temp;
            tempData.y = traits::op(temp, threadScan0[0]);
            tempData.z = traits::op(temp, threadScan0[1]);
            tempData.w = traits::op(temp, threadScan0[2]);
        }
        else
        {
            tempData.x = traits::op(temp, threadScan0[0]);
            tempData.y = traits::op(temp, threadScan0[1]);
            tempData.z = traits::op(temp, threadScan0[2]);
            tempData.w = traits::op(temp, threadScan0[3]);
        }

        int i = aiDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {                       
            outData[aiDev] = tempData; 
        }
        else 
        {       
            // we can't use vec4 because the original array isn't a multiple of 
            // 4 elements
            if ( i    < numElements) { d_out[i]   = tempData.x;
            if ((i+1) < numElements) { d_out[i+1] = tempData.y;
            if ((i+2) < numElements) { d_out[i+2] = tempData.z; } } }
        }

#ifdef DISALLOW_LOADSTORE_OVERLAP
        __syncthreads();
#endif

        temp       = s_in[bi]; 

        if (traits::isExclusive())
        {
            tempData.x = temp;
            tempData.y = traits::op(temp, threadScan1[0]);
            tempData.z = traits::op(temp, threadScan1[1]);
            tempData.w = traits::op(temp, threadScan1[2]);
        }
        else
        {
            tempData.x = traits::op(temp, threadScan1[0]);
            tempData.y = traits::op(temp, threadScan1[1]);
            tempData.z = traits::op(temp, threadScan1[2]);
            tempData.w = traits::op(temp, threadScan1[3]);
        }

        i = biDev * 4;
        if (traits::isFullBlock() || i + 3 < numElements)
        {
            outData[biDev] = tempData;
        }
        else 
        {
            // we can't use vec4 because the original array isn't a multiple of 
            // 4 elements
            if ( i    < numElements) { d_out[i]   = tempData.x;
            if ((i+1) < numElements) { d_out[i+1] = tempData.y;
            if ((i+2) < numElements) { d_out[i+2] = tempData.z; } } }
        }
    }
}

/** @brief Scan all warps of a CTA without synchronization
  * 
  * The warp-scan algorithm breaks a block of data into warp-sized chunks, and
  * scans the chunks independently with a warp of threads each.  Because warps
  * execute instructions in SIMD fashion, there is no need to synchronize in 
  * order to share data within a warp (only across warps).  Also, in SIMD the 
  * most efficient algorithm is a step-efficient algorithm.  Therefore, within
  * each warp we use a Hillis-and-Steele-style scan that takes log2(N) steps
  * to scan the warp [Daniel Hillis and Guy Steele 1986], rather than the 
  * work-efficient tree-based algorithm described by Guy Blelloch [1990] that 
  * takes 2 * log(N) steps and is in general more complex to implement.  
  * Previous versions of CUDPP used the Blelloch algorithm.  For current GPUs, 
  * the warp size is 32, so this takes five steps per warp.
  *
  * Each thread is responsible for a single element of the array to be scanned.
  * Each thread inputs a single value to the scan via \a val and returns 
  * its own scanned result element.  The threads of each warp cooperate 
  * via the shared memory array \a s_data to scan WARP_SIZE elements.
  *
  * Template parameter \a maxlevel allows this warpscan to be performed on
  * partial warps.  For example, if only the first 8 elements of each warp need
  * to be scanned, then warpscan only performs log2(8)=3 steps rather than 5.
  *
  * The computation uses 2 * WARP_SIZE elements of shared memory per warp to
  * enable warps to offset beyond their input data and receive the identity 
  * element without using any branch instructions.
  * 
  * \note s_data is declared volatile here to prevent the compiler from 
  * optimizing away writes to shared memory, and ensure correct intrawarp 
  * communication in the absence of __syncthreads.
  *
  * @return The result of the warp scan for the current thread
  * @param[in] val The current threads's input to the scan
  * @param[in,out] s_data A pointer to a temporary shared array of 2*CTA_SIZE 
  * elements used to compute the warp scans
  */
template<class T, class traits,int maxlevel>
__device__ T warpscan(T val, volatile T* s_data)
{
    // The following is the same as 2 * 32 * warpId + threadInWarp = 
    // 64*(threadIdx.x >> 5) + (threadIdx.x & (WARP_SIZE-1))
    int idx = 2 * threadIdx.x - (threadIdx.x & (WARP_SIZE-1));
    s_data[idx] = traits::identity();
    idx += WARP_SIZE;
    T t = s_data[idx] = val;  __EMUSYNC;

        // This code is needed because the warp size of device emulation
        // is only 1 thread, so sync-less cooperation within a warp doesn't 
        // work.
#ifdef __DEVICE_EMULATION__
    t = s_data[idx -  1]; __EMUSYNC; 
    s_data[idx] = traits::op(s_data[idx],t); __EMUSYNC;
    t = s_data[idx -  2]; __EMUSYNC; 
    s_data[idx] = traits::op(s_data[idx],t); __EMUSYNC;
    t = s_data[idx -  4]; __EMUSYNC; 
    s_data[idx] = traits::op(s_data[idx],t); __EMUSYNC;
    t = s_data[idx -  8]; __EMUSYNC; 
    s_data[idx] = traits::op(s_data[idx],t); __EMUSYNC;
    t = s_data[idx - 16]; __EMUSYNC; 
    s_data[idx] = traits::op(s_data[idx],t); __EMUSYNC;
#else
    if (0 <= maxlevel) { s_data[idx] = t = traits::op(t, s_data[idx - 1]); }
    if (1 <= maxlevel) { s_data[idx] = t = traits::op(t, s_data[idx - 2]); }
    if (2 <= maxlevel) { s_data[idx] = t = traits::op(t, s_data[idx - 4]); }
    if (3 <= maxlevel) { s_data[idx] = t = traits::op(t, s_data[idx - 8]); }
    if (4 <= maxlevel) { s_data[idx] = t = traits::op(t, s_data[idx -16]); }
#endif

    return s_data[idx-1];      // convert inclusive -> exclusive
}

/** @brief Perform a full CTA scan using the warp-scan algorithm
  * 
  * As described in the comment for warpscan(), the warp-scan algorithm breaks 
  * a block of data into warp-sized chunks, and scans the chunks independently 
  * with a warp of threads each.  To complete the scan, each warp <i>j</i> then 
  * writes its last element to element <i>j</i> of a temporary shared array.
  * Then a single warp exclusive-scans these "warp sums".  Finally, each thread
  * adds the result of the warp sum scan to the result of the scan from the 
  * first pass.
  *
  * Because we scan 2*CTA_SIZE elements per thread, we have to call warpscan
  * twice.
  *
  * @param x The first input value for the current thread
  * @param y The second input value for the current thread
  * @param s_data Temporary shared memory space of 2*CTA_SIZE elements for 
  * performing the scan
  */
template <class T, class traits>
__device__ void scanWarps(T x, T y, 
                          T *s_data)
{       
    T val  = warpscan<T, traits, 4>(x, s_data);
    __syncthreads(); 
    T val2 = warpscan<T, traits, 4>(y, s_data);
    
    int idx = threadIdx.x;

    if ((idx & 31)==31)
    {
        s_data[idx >> 5]                = traits::op(val, x);
        s_data[(idx + blockDim.x) >> 5] = traits::op(val2, y);
    }
    __syncthreads();

#ifndef __DEVICE_EMULATION__
    if (idx < 32)
#endif
    {
        s_data[idx] = warpscan<T,traits,(LOG_CTA_SIZE-LOG_WARP_SIZE+1)>(s_data[idx], s_data);
    }
    __syncthreads();

    val  = traits::op(val, s_data[idx >> 5]);

    val2 = traits::op(val2, s_data[(idx + blockDim.x) >> 5]);

    __syncthreads();

    s_data[idx] = val;
    s_data[idx+blockDim.x] = val2;
}

/**
* @brief CTA-level scan routine; scans s_data in shared memory in each thread block
*
* This function is the main CTA-level scan function.  It may be called by other 
* CUDA __global__ or __device__ functions. This function scans 2 * CTA_SIZE elements.
* Each thread is responsible for one element in each half of the input array.
* \note This code is intended to be run on a CTA of 128 threads.  Other sizes are
* untested.
* 
* @param[in] s_data The array to be scanned in shared memory
* @param[out] d_blockSums Array of per-block sums
* @param[in] blockSumIndex Location in \a d_blockSums to which to write this block's sum
*/
template <class T, class traits>
__device__ void scanCTA(T            *s_data, 
                        T            *d_blockSums, 
                        unsigned int blockSumIndex)
{
    T val  = s_data[threadIdx.x];
    T val2 = s_data[threadIdx.x + blockDim.x];
    __syncthreads();     

    scanWarps<T,traits>(val, val2, s_data);
    __syncthreads();  

    if (traits::writeSums() && threadIdx.x == blockDim.x - 1)
    {
        d_blockSums[blockSumIndex] = traits::op(val2, s_data[threadIdx.x + blockDim.x]);
    }
    
    
#ifdef __DEVICE_EMULATION__
    // must sync in emulation mode when doing backward scans, because otherwise the 
    // shared memory array will get reversed before the block sums are read!
    if (traits::isBackward())
        __syncthreads();
#endif
}


/** @} */ // end scan functions
/** @} */ // end cudpp_cta
