#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// CUDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision: 4400 $
// $Date: 2008-08-04 10:58:14 -0700 (Mon, 04 Aug 2008) $
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
 * @file
 * rand_md5_app.cu
 *
 * @brief CUDPP application-level rand routine for MD5
 */

#include "cudpp.h"
#include "cudpp_util.h"
#include "cudpp_plan.h"

#include <cutil.h>
#include <cstdlib>
#include <cstdio>
#include <assert.h>

#include "cta/rand_cta.cu"
#include "kernel/rand_kernel.cu"

#define RAND_CTA_SIZE 128 //128 chosen, may be changed later

/** \addtogroup cudpp_app
  *
  */

/** @name Rand Functions
 * @{
 */

/**@brief Launches the MD5 Random number generator kernel
 *
 * The MD5 Random number generator works by generating 128 bit digests which 
 * are then broken down into 32 bit chunks and stored inside \a d_out.  
 * \a d_out is expected to  be of type unsigned int and can hold \a numElements 
 * elements.
 *
 * An analysis of the stastical distribution of the MD5 random number generator
 * can be found in the original paper 
 * <a href="http://portal.acm.org/citation.cfm?id=1342263">
 * Parallel white noise generation on a GPU via cryptographic hash</a>.
 * The optimizations mentioned in the paper are also present in the CUDPP
 * version of the MD5 Random number generator.
 *
 * It is also worth pointing out that the GPU version will \b not generate 
 * the same output * as the CPU version.  This is due to the difference in the 
 * floating point accuracy and several optimizations that have been used 
 * (i.e. calculating sin using device hardware  rather than storing it in 
 * an array that the original implementation does).  However, the distribution 
 * of the numbers is well suited for random number generation, even without
 * the CPU-GPU invariance.
 *
 * @param[out] d_out the array of unsigned integers allocated on device memory
 * @param[in] seed the random seed used to vary the output
 * @param[in] numElements the number of elements in \a d_out
 * @see gen_randMD5()
 * @see cudppRand()
 * @todo: chose a better block size, perhaps a multiple of two is optimal
 */
void launchRandMD5Kernel(unsigned int * d_out, unsigned int seed, 
                         size_t numElements)
{
    //first, we need a temporary array of uints
    uint4 * dev_output;

    //figure out how many elements are needed in this array
    unsigned int devOutputsize = numElements / 4;
    devOutputsize += (numElements %4 == 0) ? 0 : 1; //used for overflow
    unsigned int memSize = devOutputsize * sizeof(uint4);


    //now figure out block size
    unsigned int blockSize = RAND_CTA_SIZE;
    if(devOutputsize < RAND_CTA_SIZE) blockSize = devOutputsize;

    unsigned int n_blocks = 
            devOutputsize/blockSize + (devOutputsize%blockSize == 0 ? 0:1);  

    //printf("Generating %u random numbers using %u blocks and %u threads per block\n", numElements, n_blocks, blockSize);
/*	old debug code now removed.
    printf("\nnumber of elements: %u, devOutputSize: %u\n", 
            numElements, devOutputsize);
    printf("number of blocks: %u blocksize: %u devOutputsize = %u\n", 
            n_blocks, blockSize, devOutputsize);
    printf("number of threads: %u\n", n_blocks * blockSize);
    printf("seed value: %u\n", seed);
*/
    //now create the memory on the device
    CUDA_SAFE_CALL( hipMalloc((void **) &dev_output, memSize));
    CUDA_SAFE_CALL( hipMemset(dev_output, 0, memSize)); 
    gen_randMD5<<<n_blocks, blockSize>>>(dev_output, devOutputsize, seed);

    //here the GPU computation is done
    //here we have all the data on the device, we copy it over into host memory


    //calculate final memSize
    //@TODO: write a template version of this which calls two different version 
    // depending if numElements %4 == 0
    size_t finalMemSize = sizeof(unsigned int) * numElements;
    CUDA_SAFE_CALL( hipMemcpy(d_out, dev_output, finalMemSize, 
                               hipMemcpyDeviceToDevice));
    CUDA_SAFE_CALL( hipFree(dev_output));
}//end launchRandMD5Kernel

#ifdef __cplusplus
extern "C"
{
#endif

/**@brief Dispatches the rand function based on the plan
 *
 * This is the dispatch call which looks at the algorithm specified in \a plan 
 * and calls the appropriate random number generation algorithm.  
 *
 * @param[out] d_out the array allocated on device memory where the random 
 * numbers will be stored
 * must be of type unsigned int
 * @param[in] numElements the number of elements in the array d_out
 * @param[in] plan pointer to CUDPPRandPlan which contains the algorithm to run
 */
void cudppRandDispatch(void * d_out, size_t numElements, 
                       const CUDPPRandPlan * plan)
{
    //switch to figure out which algorithm to run
    switch(plan->m_config.algorithm)
    {
    case CUDPP_RAND_MD5:
        //run the md5 algorithm here
        launchRandMD5Kernel( (unsigned int *) d_out, plan->m_seed, numElements);
        break;
    default:
        break;
    }//end switch

}


#ifdef __cplusplus
}
#endif
/** @} */ // end rand_app
/** @} */ // end cudpp_app
